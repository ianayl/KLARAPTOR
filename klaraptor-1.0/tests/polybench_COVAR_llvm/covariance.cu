#include "hip/hip_runtime.h"
///**
// * covariance.cu: This file is part of the PolyBench/GPU 1.0 test suite.
// *
// *
// * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
// * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
// * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
// */

#include "covariance_utils.h"

#pragma kernel_info_size_param_idx_mean_kernel = 2;
#pragma kernel_info_dim_mean_kernel = 2;

#pragma kernel_info_size_param_idx_reduce_kernel = 2;
#pragma kernel_info_dim_reduce_kernel = 2;

#pragma kernel_info_size_param_idx_covar_kernel = 2;
#pragma kernel_info_dim_covar_kernel = 2;


void
init_arrays (DATA_TYPE* data)
{
  int i, j;

  for (i = 1; i < (M + 1); i++)
    {
      for (j = 1; j < (N + 1); j++)
	{
	  data[i * (N + 1) + j] = ((DATA_TYPE) i * j) / M;
	}
    }
}

void
covariance (DATA_TYPE* data, DATA_TYPE* symmat, DATA_TYPE* mean)
{
  int i, j, j1, j2;

  /* Determine mean of column vectors of input data matrix */
  for (j = 1; j < (M + 1); j++)
    {
      mean[j] = 0.0;
      for (i = 1; i < (N + 1); i++)
	{
	  mean[j] += data[i * (M + 1) + j];
	}
      mean[j] /= FLOAT_N;
    }

  /* Center the column vectors. */
  for (i = 1; i < (N + 1); i++)
    {
      for (j = 1; j < (M + 1); j++)
	{
	  data[i * (M + 1) + j] -= mean[j];
	}
    }

  /* Calculate the m * m covariance matrix. */
  for (j1 = 1; j1 < (M + 1); j1++)
    {
      for (j2 = j1; j2 < (M + 1); j2++)
	{
	  symmat[j1 * (M + 1) + j2] = 0.0;
	  for (i = 1; i < N + 1; i++)
	    {
	      symmat[j1 * (M + 1) + j2] += data[i * (M + 1) + j1]
		  * data[i * (M + 1) + j2];
	    }
	  symmat[j2 * (M + 1) + j1] = symmat[j1 * (M + 1) + j2];
	}
    }
}

int
compareResults (DATA_TYPE* symmat, DATA_TYPE* symmat_outputFromGpu)
{
  int i, j, fail;
  fail = 0;

  for (i = 1; i < (M + 1); i++)
    {
      for (j = 1; j < (N + 1); j++)
	{
	  if (percentDiff (
	      symmat[i * (N + 1) + j],
	      symmat_outputFromGpu[i * (N + 1) + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
	    {
	      fail++;
	      return (EXIT_FAILURE);
	    }
	}
    }
  return (EXIT_SUCCESS);
//	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
//	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);

//  hipSetDevice ( GPU_DEVICE);

  return;
}

__global__ void
mean_kernel (DATA_TYPE *mean, DATA_TYPE *data, int M, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

  if ((j >= 1) && (j < (M + 1)))
    {
      mean[j] = 0.0;

      int i;
      for (i = 1; i < (N + 1); i++)
	{
	  mean[j] += data[i * (M + 1) + j];
	}
      mean[j] /= (DATA_TYPE) FLOAT_N;
    }
}

__global__ void
reduce_kernel (DATA_TYPE *mean, DATA_TYPE *data, int M, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int i = blockIdx.y * blockDim.y + threadIdx.y + 1;

  if ((i >= 1) && (i < (N + 1)) && (j >= 1) && (j < (M + 1)))
    {
      data[i * (M + 1) + j] -= mean[j];
    }
}

__global__ void
covar_kernel (DATA_TYPE *symmat, DATA_TYPE *data, int M, int N)
{
  int j1 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int i, j2;

  if ((j1 >= 1) && (j1 < (M + 1)))
    {
      for (j2 = j1; j2 < (M + 1); j2++)
	{
	  symmat[j1 * (M + 1) + j2] = 0.0;
	  for (i = 1; i < (N + 1); i++)
	    {
	      symmat[j1 * (M + 1) + j2] += data[i * (M + 1) + j1]
		  * data[i * (M + 1) + j2];
	    }
	  symmat[j2 * (M + 1) + j1] = symmat[j1 * (M + 1) + j2];
	}
    }
}

void
covarianceCuda (DATA_TYPE* data, DATA_TYPE* symmat, DATA_TYPE* mean,
		DATA_TYPE* symmat_outputFromGpu)
{
  double t_start, t_end;

  DATA_TYPE *data_gpu;
  DATA_TYPE *mean_gpu;
  DATA_TYPE *symmat_gpu;

  hipMalloc ((void **) &data_gpu, sizeof(DATA_TYPE) * (M + 1) * (N + 1));
  hipMalloc ((void **) &symmat_gpu, sizeof(DATA_TYPE) * (M + 1) * (M + 1));
  hipMalloc ((void **) &mean_gpu, sizeof(DATA_TYPE) * (M + 1));
  hipMemcpy (data_gpu, data, sizeof(DATA_TYPE) * (M + 1) * (N + 1),
	      hipMemcpyHostToDevice);
  hipMemcpy (symmat_gpu, symmat, sizeof(DATA_TYPE) * (M + 1) * (M + 1),
	      hipMemcpyHostToDevice);
  hipMemcpy (mean_gpu, mean, sizeof(DATA_TYPE) * (M + 1),
	      hipMemcpyHostToDevice);

  dim3 block1 (DIM_THREAD_BLOCK_KERNEL_1_X, DIM_THREAD_BLOCK_KERNEL_1_Y);
  dim3 grid1 (
      (size_t) (ceil ((float) M) / ((float) DIM_THREAD_BLOCK_KERNEL_1_X)), 1);

  dim3 block2 (DIM_THREAD_BLOCK_KERNEL_2_X, DIM_THREAD_BLOCK_KERNEL_2_Y);
  dim3 grid2 (
      (size_t) (ceil ((float) M) / ((float) DIM_THREAD_BLOCK_KERNEL_2_X)),
      (size_t) (ceil ((float) N) / ((float) DIM_THREAD_BLOCK_KERNEL_2_X)));

  dim3 block3 (DIM_THREAD_BLOCK_KERNEL_3_X, DIM_THREAD_BLOCK_KERNEL_3_Y);
  dim3 grid3 (
      (size_t) (ceil ((float) M) / ((float) DIM_THREAD_BLOCK_KERNEL_3_X)), 1);

//	t_start = rtclock();

  cuda_timer t1, t2, t3;

  cuda_timer_init (t1);
  cuda_timer_init (t2);
  cuda_timer_init (t3);

  cuda_timer_record_start (t1);
  mean_kernel <<<grid1, block1>>> (mean_gpu, data_gpu, M, N);
  cudaCheckKernel ()
  ;
  cuda_timer_record_stop (t1);
  hipDeviceSynchronize ();

  cuda_timer_record_start (t2);
  reduce_kernel <<<grid2, block2>>> (mean_gpu, data_gpu, M, N);
  cudaCheckKernel ()
  ;
  cuda_timer_record_stop (t2);
  hipDeviceSynchronize ();

  cuda_timer_record_start (t3);
  covar_kernel <<<grid3, block3>>> (symmat_gpu, data_gpu, M, N);
  cudaCheckKernel ()
  ;
  cuda_timer_record_stop (t3);
  hipDeviceSynchronize ();

  hipMemcpy (symmat_outputFromGpu, symmat_gpu,
	      sizeof(DATA_TYPE) * (M + 1) * (N + 1), hipMemcpyDeviceToHost);

  cuda_timer_record_get_elapsed_time(t1);
  cuda_timer_record_get_elapsed_time(t2);
  cuda_timer_record_get_elapsed_time(t3);
  printf ("[trace: n=%d, bx=%d, by=%d, "
	  "elapsed_mean_kernel=%0.4f (ms),\n"
	  " elapsed_reduce_kernel=%0.4f (ms), "
	  " elapsed_covar_kernel=%0.4f (ms)] ... ",
	  M, DIM_THREAD_BLOCK_KERNEL_3_X, DIM_THREAD_BLOCK_KERNEL_3_Y,
	  t1.elapsed_time, t2.elapsed_time, t3.elapsed_time);
  hipFree (data_gpu);
  hipFree (symmat_gpu);
  hipFree (mean_gpu);
  cuda_timer_destroy (t1);
  cuda_timer_destroy (t2);
  cuda_timer_destroy (t3);
}

int
main (int argc, char** argv)
{
//  double t_start, t_end;

  DATA_TYPE* data;
  DATA_TYPE* symmat;
  DATA_TYPE* mean;
  DATA_TYPE* symmat_outputFromGpu;

  int n = 256, bx = 32, by = 8, n_repeat=1, verify=0;

  bx=DIM_THREAD_BLOCK_KERNEL_1_X;
  by=DIM_THREAD_BLOCK_KERNEL_1_Y;

  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);
  if (argc>4)
  	n_repeat=atoi(argv[4]);
	if (argc>5)
  	verify=atoi(argv[5]);

  M = N = n;
  DIM_THREAD_BLOCK_KERNEL_1_X = bx;
  DIM_THREAD_BLOCK_KERNEL_1_Y = by;
  DIM_THREAD_BLOCK_KERNEL_2_X = bx;
  DIM_THREAD_BLOCK_KERNEL_2_Y = by;
  DIM_THREAD_BLOCK_KERNEL_3_X = bx;
  DIM_THREAD_BLOCK_KERNEL_3_Y = by;

  data = (DATA_TYPE*) malloc ((M + 1) * (N + 1) * sizeof(DATA_TYPE));
  symmat = (DATA_TYPE*) malloc ((M + 1) * (M + 1) * sizeof(DATA_TYPE));
  mean = (DATA_TYPE*) malloc ((M + 1) * sizeof(DATA_TYPE));
  symmat_outputFromGpu = (DATA_TYPE*) malloc (
      (M + 1) * (M + 1) * sizeof(DATA_TYPE));

  init_arrays (data);

  GPU_argv_init ();

	for(int i=0;i<n_repeat;i++)
	  covarianceCuda (data, symmat, mean, symmat_outputFromGpu);
	
	if(verify)
	{
//  t_start = rtclock ();
	for(int i=0;i<n_repeat;i++)
	  covariance (data, symmat, mean);
//  t_end = rtclock ();
//  fprintf (stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

  int s = compareResults (symmat, symmat_outputFromGpu);
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");
	}
  free (data);
  free (symmat);
  free (mean);
  free (symmat_outputFromGpu);

  return 0;
}

