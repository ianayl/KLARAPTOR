#include "hip/hip_runtime.h"
///**
// * atax.cu: This file is part of the PolyBench/GPU 1.0 test suite.
// *
// *
// * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
// * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
// * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
// */
//

#include "atax_utils.h"

#pragma kernel_info_size_param_idx_atax_kernel1 = 3;
#pragma kernel_info_dim_atax_kernel1 = 2;

#pragma kernel_info_size_param_idx_atax_kernel2 = 3;
#pragma kernel_info_dim_atax_kernel2 = 2;

void
init_array (DATA_TYPE *x, DATA_TYPE *A)
{
  int i, j;

  for (i = 0; i < NX; i++)
    {
      x[i] = i * M_PI;
      for (j = 0; j < NY; j++)
	{
	  A[i * NY + j] = ((DATA_TYPE) i * (j)) / NX;
	}
    }
}

int
compareResults (DATA_TYPE *z, DATA_TYPE *z_outputFromGpu)
{
  int i, fail;
  fail = 0;

  for (i = 0; i < NY; i++)
    {
      if (percentDiff (z[i], z_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
	{
	  fail++;
	  return (EXIT_FAILURE);
	}
    }

  // print results
//  printf (
//      "Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n",
//      PERCENT_DIFF_ERROR_THRESHOLD,
//      fail);
  return (EXIT_SUCCESS);
}

void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
//  printf ("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
//  hipSetDevice ( GPU_DEVICE);
}

__global__ void
atax_kernel1 (DATA_TYPE *A, DATA_TYPE *x, DATA_TYPE *tmp, int NX, int NY)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < NX)
    {
      int j;
      for (j = 0; j < NY; j++)
	{
	  tmp[i] += A[i * NY + j] * x[j];
	}
    }
}

__global__ void
atax_kernel2 (DATA_TYPE *A, DATA_TYPE *y, DATA_TYPE *tmp, int NX, int NY)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (j < NY)
    {
      int i;
      for (i = 0; i < NX; i++)
	{
	  y[j] += A[i * NY + j] * tmp[i];
	}
    }
}

void
atax_cpu (DATA_TYPE* A, DATA_TYPE* x, DATA_TYPE* y, DATA_TYPE* tmp)
{
  int i, j;

  for (i = 0; i < NY; i++)
    {
      y[i] = 0;
    }

  for (i = 0; i < NX; i++)
    {
      tmp[i] = 0;

      for (j = 0; j < NY; j++)
	{
	  tmp[i] = tmp[i] + A[i * NY + j] * x[j];
	}

      for (j = 0; j < NY; j++)
	{
	  y[j] = y[j] + A[i * NY + j] * tmp[i];
	}
    }
}

void
ataxGpu (DATA_TYPE* A, DATA_TYPE* x, DATA_TYPE* y, DATA_TYPE* tmp,
	 DATA_TYPE* y_outputFromGpu)
{
  double t_start, t_end;

  DATA_TYPE *A_gpu;
  DATA_TYPE *x_gpu;
  DATA_TYPE *y_gpu;
  DATA_TYPE *tmp_gpu;

  hipMalloc ((void **) &A_gpu, sizeof(DATA_TYPE) * NX * NY);
  hipMalloc ((void **) &x_gpu, sizeof(DATA_TYPE) * NY);
  hipMalloc ((void **) &y_gpu, sizeof(DATA_TYPE) * NY);
  hipMalloc ((void **) &tmp_gpu, sizeof(DATA_TYPE) * NX);

  hipMemcpy (A_gpu, A, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
  hipMemcpy (x_gpu, x, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
  hipMemcpy (y_gpu, y, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
  hipMemcpy (tmp_gpu, tmp, sizeof(DATA_TYPE) * NX, hipMemcpyHostToDevice);

  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid1 ((size_t) (ceil (((float) NX) / ((float) block.x))), 1);
  dim3 grid2 ((size_t) (ceil (((float) NY) / ((float) block.x))), 1);

//	t_start = rtclock();

  cuda_timer t1, t2;
  cuda_timer_init (t1);
  cuda_timer_init (t2);

  cuda_timer_record_start (t1);
  atax_kernel1 <<<grid1, block>>> (A_gpu, x_gpu, tmp_gpu, NX, NY);
  cudaCheckKernel ()
  ;
  cuda_timer_record_stop (t1);
  hipDeviceSynchronize ();

  cuda_timer_record_start (t2);
  atax_kernel2 <<<grid2, block>>> (A_gpu, y_gpu, tmp_gpu, NX, NY);
  cudaCheckKernel ()
  ;
  cuda_timer_record_stop (t2);
  hipDeviceSynchronize ();
//	t_end = rtclock();
//	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

  hipMemcpy (y_outputFromGpu, y_gpu, sizeof(DATA_TYPE) * NX,
	      hipMemcpyDeviceToHost);

  cuda_timer_record_get_elapsed_time (t1);
  cuda_timer_record_get_elapsed_time (t2);
  printf ("[trace: n=%d, bx=%d, by=%d, "
	  "elapsed_atax_kernel1=%0.4f (ms), "
	  "elapsed_atax_kernel2=%0.4f (ms)] ... \n",
	  NX, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t1.elapsed_time,
	  t2.elapsed_time);

  hipFree (A_gpu);
  hipFree (x_gpu);
  hipFree (y_gpu);
  hipFree (tmp_gpu);
  cuda_timer_destroy (t1);
  cuda_timer_destroy (t2);
}

int
main (int argc, char** argv)
{
//	double t_start, t_end;

  int n = 256, bx = 32, by = 8, n_repeat=1, verify=0;


  bx=DIM_THREAD_BLOCK_X;
  by=DIM_THREAD_BLOCK_Y;

  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);
  if (argc > 4)
    n_repeat = atoi (argv[4]);
	if (argc > 5)
    verify = atoi (argv[5]);

  NX = NY = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;

  DATA_TYPE* A;
  DATA_TYPE* x;
  DATA_TYPE* y;
  DATA_TYPE* y_outputFromGpu;
  DATA_TYPE* tmp;

  A = (DATA_TYPE*) malloc (NX * NY * sizeof(DATA_TYPE));
  x = (DATA_TYPE*) malloc (NY * sizeof(DATA_TYPE));
  y = (DATA_TYPE*) malloc (NY * sizeof(DATA_TYPE));
  y_outputFromGpu = (DATA_TYPE*) malloc (NY * sizeof(DATA_TYPE));
  tmp = (DATA_TYPE*) malloc (NX * sizeof(DATA_TYPE));

  init_array (x, A);

  GPU_argv_init ();
	
	for(int i=0;i<n_repeat;i++)
	  ataxGpu (A, x, y, tmp, y_outputFromGpu);

	if(verify)
	{
//  t_start = rtclock ();
	for(int i=0;i<n_repeat;i++)
  	atax_cpu (A, x, y, tmp);
//  t_end = rtclock ();
//  fprintf (stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

  int s = compareResults (y, y_outputFromGpu);
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");
	}
  free (A);
  free (x);
  free (y);
  free (y_outputFromGpu);
  free (tmp);

  return 0;
}

