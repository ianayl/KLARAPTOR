#include "hip/hip_runtime.h"
///**
// * bicg.cu: This file is part of the PolyBench/GPU 1.0 test suite.
// *
// *
// * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
// * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
// * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
// */

#include "bicg_utils.h"

#pragma kernel_info_size_param_idx_bicg_kernel1 = 3;
#pragma kernel_info_dim_bicg_kernel1 = 2;

#pragma kernel_info_size_param_idx_bicg_kernel2 = 3;
#pragma kernel_info_dim_bicg_kernel2 = 2;

void
init_array (DATA_TYPE *A, DATA_TYPE *p, DATA_TYPE *r)
{
  int i, j;

  for (i = 0; i < NX; i++)
    {
      r[i] = i * M_PI;

      for (j = 0; j < NY; j++)
	{
	  A[i * NY + j] = ((DATA_TYPE) i * j) / NX;
	}
    }

  for (i = 0; i < NY; i++)
    {
      p[i] = i * M_PI;
    }
}

int
compareResults (DATA_TYPE* s, DATA_TYPE* s_outputFromGpu, DATA_TYPE* q,
		DATA_TYPE* q_outputFromGpu)
{
  int i, fail;
  fail = 0;

  // Compare s with s_cuda
  for (i = 0; i < NX; i++)
    {
      if (percentDiff (q[i], q_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
	{
	  fail++;
	  return (EXIT_FAILURE);
	}
    }

  for (i = 0; i < NY; i++)
    {
      if (percentDiff (s[i], s_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
	{
	  fail++;
	  return (EXIT_FAILURE);
	}
    }

  return (EXIT_SUCCESS);
  // print results
//  printf (
//      "Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n",
//      PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
//  printf ("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
//  hipSetDevice ( GPU_DEVICE);
}

//Distributed (split) from initial loop and permuted into reverse order to allow parallelism...
__global__ void
bicg_kernel1 (DATA_TYPE *A, DATA_TYPE *r, DATA_TYPE *s, int NX, int NY)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (j < NY)
    {
      s[j] = 0.0f;

      int i;
      for (i = 0; i < NX; i++)
	{
	  s[j] += A[i * NY + j] * r[i];
	}
    }
}

//Distributed (split) from initial loop to allow parallelism
__global__ void
bicg_kernel2 (DATA_TYPE *A, DATA_TYPE *p, DATA_TYPE *q, int NX, int NY)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < NX)
    {
      q[i] = 0.0f;

      int j;
      for (j = 0; j < NY; j++)
	{
	  q[i] += A[i * NY + j] * p[j];
	}
    }
}

void
bicg_cpu (DATA_TYPE* A, DATA_TYPE* r, DATA_TYPE* s, DATA_TYPE* p, DATA_TYPE* q)
{
  int i, j;

  for (i = 0; i < NY; i++)
    {
      s[i] = 0.0;
    }

  for (i = 0; i < NX; i++)
    {
      q[i] = 0.0;
      for (j = 0; j < NY; j++)
	{
	  s[j] = s[j] + r[i] * A[i * NY + j];
	  q[i] = q[i] + A[i * NY + j] * p[j];
	}
    }
}

void
bicgCuda (DATA_TYPE* A, DATA_TYPE* r, DATA_TYPE* s, DATA_TYPE* p, DATA_TYPE* q,
	  DATA_TYPE* s_outputFromGpu, DATA_TYPE* q_outputFromGpu)
{
  double t_start, t_end;

  DATA_TYPE *A_gpu;
  DATA_TYPE *q_gpu;
  DATA_TYPE *p_gpu;
  DATA_TYPE *r_gpu;
  DATA_TYPE *s_gpu;

  hipMalloc ((void **) &A_gpu, sizeof(DATA_TYPE) * NX * NY);
  hipMalloc ((void **) &r_gpu, sizeof(DATA_TYPE) * NX);
  hipMalloc ((void **) &s_gpu, sizeof(DATA_TYPE) * NY);
  hipMalloc ((void **) &p_gpu, sizeof(DATA_TYPE) * NY);
  hipMalloc ((void **) &q_gpu, sizeof(DATA_TYPE) * NX);
  hipMemcpy (A_gpu, A, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
  hipMemcpy (r_gpu, r, sizeof(DATA_TYPE) * NX, hipMemcpyHostToDevice);
  hipMemcpy (s_gpu, s, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
  hipMemcpy (p_gpu, p, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
  hipMemcpy (q_gpu, q, sizeof(DATA_TYPE) * NX, hipMemcpyHostToDevice);

  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid1 ((size_t) (ceil (((float) NY) / ((float) block.x))), 1);
  dim3 grid2 ((size_t) (ceil (((float) NX) / ((float) block.x))), 1);

  cuda_timer t1, t2;
  cuda_timer_init (t1);
  cuda_timer_init (t2);
  cuda_timer_record_start (t1);
//  t_start = rtclock ();
  bicg_kernel1 <<<grid1, block>>> (A_gpu, r_gpu, s_gpu, NX, NY);
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t1);
  hipDeviceSynchronize ();

  cuda_timer_record_start (t2);
  bicg_kernel2 <<<grid2, block>>> (A_gpu, p_gpu, q_gpu, NX, NY);
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t2);
  hipDeviceSynchronize ();
//  t_end = rtclock ();
//  fprintf (stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

  cuda_timer_record_get_elapsed_time (t1);
  cuda_timer_record_get_elapsed_time (t2);

  hipMemcpy (s_outputFromGpu, s_gpu, sizeof(DATA_TYPE) * NY,
	      hipMemcpyDeviceToHost);
  hipMemcpy (q_outputFromGpu, q_gpu, sizeof(DATA_TYPE) * NX,
	      hipMemcpyDeviceToHost);

  printf ("[trace: n=%d, bx=%d, by=%d, "
	  "elapsed_bicg_kernel1=%0.4f (ms), "
	  "elapsed_bicg_kernel2=%0.4f (ms)] ... ",
	  NX, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t1.elapsed_time,
	  t2.elapsed_time);

  hipFree (A_gpu);
  hipFree (r_gpu);
  hipFree (s_gpu);
  hipFree (p_gpu);
  hipFree (q_gpu);

  cuda_timer_destroy (t1);
  cuda_timer_destroy (t2);
}

int
main (int argc, char** argv)
{
//  double t_start, t_end;
  int n = 256, bx = 32, by = 8, n_repeat=1, verify=0;

  bx=DIM_THREAD_BLOCK_X;
  by=DIM_THREAD_BLOCK_Y;

  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);
  if (argc>4)
  	n_repeat=atoi(argv[4]);
	if (argc > 5)
    verify = atoi (argv[5]);

  NX = NY = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;

  DATA_TYPE* A;
  DATA_TYPE* r;
  DATA_TYPE* s;
  DATA_TYPE* p;
  DATA_TYPE* q;
  DATA_TYPE* s_outputFromGpu;
  DATA_TYPE* q_outputFromGpu;

  A = (DATA_TYPE*) malloc (NX * NY * sizeof(DATA_TYPE));
  r = (DATA_TYPE*) malloc (NX * sizeof(DATA_TYPE));
  s = (DATA_TYPE*) malloc (NY * sizeof(DATA_TYPE));
  p = (DATA_TYPE*) malloc (NY * sizeof(DATA_TYPE));
  q = (DATA_TYPE*) malloc (NX * sizeof(DATA_TYPE));
  s_outputFromGpu = (DATA_TYPE*) malloc (NY * sizeof(DATA_TYPE));
  q_outputFromGpu = (DATA_TYPE*) malloc (NX * sizeof(DATA_TYPE));

  init_array (A, p, r);

  GPU_argv_init ();
	
	for(int i=0;i<n_repeat;i++)
	  bicgCuda (A, r, s, p, q, s_outputFromGpu, q_outputFromGpu);
	if(verify)
	{
//  t_start = rtclock ();
  for(int i=0;i<n_repeat;i++)	
	  bicg_cpu (A, r, s, p, q);
//  t_end = rtclock ();

//  fprintf (stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

  int stat = compareResults (s, s_outputFromGpu, q, q_outputFromGpu);

  if (stat == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");
	}
  free (A);
  free (r);
  free (s);
  free (p);
  free (q);
  free (s_outputFromGpu);
  free (q_outputFromGpu);

  return 0;
}

