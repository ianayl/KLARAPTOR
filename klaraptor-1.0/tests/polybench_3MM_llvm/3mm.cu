#include "hip/hip_runtime.h"
///**
// * 3mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
// *
// *
// * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
// * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
// * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
// */

#include "3mm_utils.h"

#pragma kernel_info_size_param_idx_mm3_kernel1 = 3;
#pragma kernel_info_dim_mm3_kernel1 = 2;

#pragma kernel_info_size_param_idx_mm3_kernel2 = 3;
#pragma kernel_info_dim_mm3_kernel2 = 2;

#pragma kernel_info_size_param_idx_mm3_kernel3 = 3;
#pragma kernel_info_dim_mm3_kernel3 = 2;

void
init_array (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D)
{
  int i, j;

  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NK; j++)
	{
	  A[i * NK + j] = ((DATA_TYPE) i * j) / NI;
	}
    }

  for (i = 0; i < NK; i++)
    {
      for (j = 0; j < NJ; j++)
	{
	  B[i * NJ + j] = ((DATA_TYPE) i * (j + 1)) / NJ;
	}
    }

  for (i = 0; i < NJ; i++)
    {
      for (j = 0; j < NM; j++)
	{
	  C[i * NM + j] = ((DATA_TYPE) i * (j + 3)) / NL;
	}
    }

  for (i = 0; i < NM; i++)
    {
      for (j = 0; j < NL; j++)
	{
	  D[i * NL + j] = ((DATA_TYPE) i * (j + 2)) / NK;
	}
    }
}

int
compareResults (DATA_TYPE *G, DATA_TYPE *G_outputFromGpu)
{
  int i, j, fail;
  fail = 0;

  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NL; j++)
	{
	  if (percentDiff (
	      G[i * NL + j],
	      G_outputFromGpu[i * NL + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
	    {
	      fail++;
	      return (EXIT_FAILURE);
	    }
	}
    }

  // print results
//  printf (
//      "Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n",
//      PERCENT_DIFF_ERROR_THRESHOLD, fail);
  return (EXIT_SUCCESS);
}

void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
//  printf ("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
//  hipSetDevice ( GPU_DEVICE);
}

__global__ void
mm3_kernel1 (DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *E, int NI, int NJ, int NK)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < NI) && (j < NJ))
    {
      int k;
      for (k = 0; k < NK; k++)
	{
	  E[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
	}
    }
}

__global__ void
mm3_kernel2 (DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *F, int NJ, int NL, int NM)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < NJ) && (j < NL))
    {
      int k;
      for (k = 0; k < NM; k++)
	{
	  F[i * NL + j] += C[i * NM + k] * D[k * NL + j];
	}
    }
}

__global__ void
mm3_kernel3 (DATA_TYPE *E, DATA_TYPE *F, DATA_TYPE *G, int NI, int NJ, int NL)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < NI) && (j < NL))
    {
      int k;
      for (k = 0; k < NJ; k++)
	{
	  G[i * NL + j] += E[i * NJ + k] * F[k * NL + j];
	}
    }
}

void
mm3_cpu (DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *E,
	 DATA_TYPE *F, DATA_TYPE *G)
{
  int i, j, k;

  /* E := A*B */
  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NJ; j++)
	{
	  E[i * NJ + j] = 0;
	  for (k = 0; k < NK; ++k)
	    {
	      E[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
	    }
	}
    }

  /* F := C*D */
  for (i = 0; i < NJ; i++)
    {
      for (j = 0; j < NL; j++)
	{
	  F[i * NL + j] = 0;
	  for (k = 0; k < NM; ++k)
	    {
	      F[i * NL + j] += C[i * NM + k] * D[k * NL + j];
	    }
	}
    }

  /* G := E*F */
  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NL; j++)
	{
	  G[i * NL + j] = 0;
	  for (k = 0; k < NJ; ++k)
	    {
	      G[i * NL + j] += E[i * NJ + k] * F[k * NL + j];
	    }
	}
    }
}

void
mm3Cuda (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E,
	 DATA_TYPE* F, DATA_TYPE* G, DATA_TYPE* G_outputFromGpu)
{
  double t_start, t_end;

  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;
  DATA_TYPE *C_gpu;
  DATA_TYPE *D_gpu;
  DATA_TYPE *E_gpu;
  DATA_TYPE *F_gpu;
  DATA_TYPE *G_gpu;

  hipMalloc ((void **) &A_gpu, sizeof(DATA_TYPE) * NI * NK);
  hipMalloc ((void **) &B_gpu, sizeof(DATA_TYPE) * NK * NJ);
  hipMalloc ((void **) &C_gpu, sizeof(DATA_TYPE) * NJ * NM);
  hipMalloc ((void **) &D_gpu, sizeof(DATA_TYPE) * NM * NL);
  hipMalloc ((void **) &E_gpu, sizeof(DATA_TYPE) * NI * NJ);
  hipMalloc ((void **) &F_gpu, sizeof(DATA_TYPE) * NJ * NL);
  hipMalloc ((void **) &G_gpu, sizeof(DATA_TYPE) * NI * NL);

  hipMemcpy (A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
  hipMemcpy (B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
  hipMemcpy (C_gpu, C, sizeof(DATA_TYPE) * NJ * NM, hipMemcpyHostToDevice);
  hipMemcpy (D_gpu, D, sizeof(DATA_TYPE) * NM * NL, hipMemcpyHostToDevice);
  hipMemcpy (E_gpu, E, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
  hipMemcpy (F_gpu, F, sizeof(DATA_TYPE) * NJ * NL, hipMemcpyHostToDevice);
  hipMemcpy (G_gpu, G, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);

  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid1 ((size_t) (ceil (((float) NJ) / ((float) DIM_THREAD_BLOCK_X))),
	      (size_t) (ceil ((float) NI / ((float) DIM_THREAD_BLOCK_Y))));
  dim3 grid2 ((size_t) (ceil (((float) NL) / ((float) DIM_THREAD_BLOCK_X))),
	      (size_t) (ceil ((float) NJ / ((float) DIM_THREAD_BLOCK_Y))));
  dim3 grid3 ((size_t) (ceil (((float) NL) / ((float) DIM_THREAD_BLOCK_X))),
	      (size_t) (ceil ((float) NI / ((float) DIM_THREAD_BLOCK_Y))));

  cuda_timer t1, t2, t3;

  cuda_timer_init (t1);
  cuda_timer_init (t2);
  cuda_timer_init (t3);

//  t_start = rtclock ();
  cuda_timer_record_start (t1);
  mm3_kernel1 <<<grid1, block>>> (A_gpu, B_gpu, E_gpu, NI, NJ, NK);
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t1);
  hipDeviceSynchronize ();

  cuda_timer_record_start (t2);
  mm3_kernel2 <<<grid2, block>>> (C_gpu, D_gpu, F_gpu, NJ, NL, NM);
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t2);
  hipDeviceSynchronize ();

  cuda_timer_record_start (t3);
  mm3_kernel3 <<<grid3, block>>> (E_gpu, F_gpu, G_gpu, NI, NJ, NL);
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t3);
  hipDeviceSynchronize ();
//  t_end = rtclock ();
  hipMemcpy (G_outputFromGpu, G_gpu, sizeof(DATA_TYPE) * NI * NL,
	      hipMemcpyDeviceToHost);

//  fprintf (stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

 cuda_timer_record_get_elapsed_time(t1);
 cuda_timer_record_get_elapsed_time(t2);
 cuda_timer_record_get_elapsed_time(t3);

  printf (
        "[trace: n=%d, bx=%d, by=%d, "
        "elapsed_mm3_kernel1=%0.4f (ms), "
        "elapsed_mm3_kernel2=%0.4f (ms), "
        "elapsed_mm3_kernel3=%0.4f (ms)] ... ",
        NI, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y,
        t1.elapsed_time,
        t2.elapsed_time,
        t3.elapsed_time);

  hipFree (A_gpu);
  hipFree (B_gpu);
  hipFree (C_gpu);
  hipFree (D_gpu);
  hipFree (E_gpu);
  hipFree (F_gpu);
  hipFree (G_gpu);
}

int
main (int argc, char** argv)
{
  int n = 256, bx = 32, by = 8, n_repeat=1, verify=0;

  bx=DIM_THREAD_BLOCK_X;
  by=DIM_THREAD_BLOCK_Y;

  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);
  if (argc>4)
  	n_repeat=atoi(argv[4]);
	if (argc > 5)
    verify = atoi (argv[5]);

  NI = NJ = NK = NL = NM = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;

//	double t_start, t_end;

  DATA_TYPE* A;
  DATA_TYPE* B;
  DATA_TYPE* C;
  DATA_TYPE* D;
  DATA_TYPE* E;
  DATA_TYPE* F;
  DATA_TYPE* G;
  DATA_TYPE* G_outputFromGpu;

  A = (DATA_TYPE*) malloc (NI * NK * sizeof(DATA_TYPE));
  B = (DATA_TYPE*) malloc (NK * NJ * sizeof(DATA_TYPE));
  C = (DATA_TYPE*) malloc (NJ * NM * sizeof(DATA_TYPE));
  D = (DATA_TYPE*) malloc (NM * NL * sizeof(DATA_TYPE));
  E = (DATA_TYPE*) malloc (NI * NJ * sizeof(DATA_TYPE));
  F = (DATA_TYPE*) malloc (NJ * NL * sizeof(DATA_TYPE));
  G = (DATA_TYPE*) malloc (NI * NL * sizeof(DATA_TYPE));
  G_outputFromGpu = (DATA_TYPE*) malloc (NI * NL * sizeof(DATA_TYPE));

  init_array (A, B, C, D);

  GPU_argv_init ();

	for(int i=0;i<n_repeat;i++)
  	mm3Cuda (A, B, C, D, E, F, G, G_outputFromGpu);

	if(verify)
	{
//  t_start = rtclock ();
	for(int i=0;i<n_repeat;i++)
	  mm3_cpu (A, B, C, D, E, F, G);

//  t_end = rtclock ();

//  fprintf (stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

  int s = compareResults (G, G_outputFromGpu);
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");
	}
  free (A);
  free (B);
  free (C);
  free (D);
  free (E);
  free (F);
  free (G);
  free (G_outputFromGpu);

  return 0;
}

