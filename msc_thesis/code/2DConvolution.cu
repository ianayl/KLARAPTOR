#include "hip/hip_runtime.h"
///**
// * 2DConvolution.cu: This file is part of the PolyBench/GPU 1.0 test suite.
// *
// *
// * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
// * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
// * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
// */

#include "2dconv_utils.h"


#pragma kernel_info_size_param_idx_Convolution2D_kernel = 2;
#pragma kernel_info_dim_Convolution2D_kernel = 2;

void
conv2D (DATA_TYPE* A, DATA_TYPE* B)
{
  int i, j;
  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

  c11 = +0.2;
  c21 = +0.5;
  c31 = -0.8;
  c12 = -0.3;
  c22 = +0.6;
  c32 = -0.9;
  c13 = +0.4;
  c23 = +0.7;
  c33 = +0.10;

  for (i = 1; i < NI - 1; ++i) // 0
    {
      for (j = 1; j < NJ - 1; ++j) // 1
	{
	  B[i * NJ + j] = c11 * A[(i - 1) * NJ + (j - 1)]
	      + c12 * A[(i + 0) * NJ + (j - 1)]
	      + c13 * A[(i + 1) * NJ + (j - 1)]
	      + c21 * A[(i - 1) * NJ + (j + 0)]
	      + c22 * A[(i + 0) * NJ + (j + 0)]
	      + c23 * A[(i + 1) * NJ + (j + 0)]
	      + c31 * A[(i - 1) * NJ + (j + 1)]
	      + c32 * A[(i + 0) * NJ + (j + 1)]
	      + c33 * A[(i + 1) * NJ + (j + 1)];
	}
    }
}

void
init (DATA_TYPE* A)
{
  int i, j;

  for (i = 0; i < NI; ++i)
    {
      for (j = 0; j < NJ; ++j)
	{
	  A[i * NJ + j] = (float) rand () / RAND_MAX;
	}
    }
}

int
compareResults (DATA_TYPE* B, DATA_TYPE* B_outputFromGpu)
{
  int i, j, fail;
  fail = 0;

  // Compare a and b
  for (i = 1; i < (NI - 1); i++)
    {
      for (j = 1; j < (NJ - 1); j++)
	{
	  if (percentDiff (
	      B[i * NJ + j],
	      B_outputFromGpu[i * NJ + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
	    {
	      fail++;
	      return (EXIT_FAILURE);
	    }
	}
    }

  // Print results
//  printf (
//      "Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n",
//      PERCENT_DIFF_ERROR_THRESHOLD,
//      fail);
  return (EXIT_SUCCESS);

}

void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
//  printf ("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);
}

__global__ void
Convolution2D_kernel (DATA_TYPE *A, DATA_TYPE *B, int NI, int NJ)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

  c11 = +0.2;
  c21 = +0.5;
  c31 = -0.8;
  c12 = -0.3;
  c22 = +0.6;
  c32 = -0.9;
  c13 = +0.4;
  c23 = +0.7;
  c33 = +0.10;

  if ((i < NI - 1) && (j < NJ - 1) && (i > 0) && (j > 0))
    {
      B[i * NJ + j] = c11 * A[(i - 1) * NJ + (j - 1)]
	  + c21 * A[(i - 1) * NJ + (j + 0)] + c31 * A[(i - 1) * NJ + (j + 1)]
	  + c12 * A[(i + 0) * NJ + (j - 1)] + c22 * A[(i + 0) * NJ + (j + 0)]
	  + c32 * A[(i + 0) * NJ + (j + 1)] + c13 * A[(i + 1) * NJ + (j - 1)]
	  + c23 * A[(i + 1) * NJ + (j + 0)] + c33 * A[(i + 1) * NJ + (j + 1)];
    }
}

void
convolution2DCuda (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* B_outputFromGpu)
{

  cuda_timer t_conv;
  cuda_timer_init (t_conv);

  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;

  hipMalloc ((void **) &A_gpu, sizeof(DATA_TYPE) * NI * NJ);
  hipMalloc ((void **) &B_gpu, sizeof(DATA_TYPE) * NI * NJ);
  hipMemcpy (A_gpu, A, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);

  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid ((size_t) ceil (((float) NI) / ((float) block.x)),
	     (size_t) ceil (((float) NJ) / ((float) block.y)));

  cuda_timer_record_start (t_conv);
  Convolution2D_kernel <<<grid, block>>> (A_gpu, B_gpu, NI, NJ);
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t_conv);
  hipDeviceSynchronize ();
  hipMemcpy (B_outputFromGpu, B_gpu, sizeof(DATA_TYPE) * NI * NJ,
	      hipMemcpyDeviceToHost);
  hipFree (A_gpu);
  hipFree (B_gpu);

  cuda_timer_record_get_elapsed_time (t_conv);

  printf (
      "[trace: n=%d, bx=%d, by=%d, elapsed_Convolution2D_kernel=%0.4f (ms)] ... ",
      NI, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t_conv.elapsed_time);
}

int
main (int argc, char **argv)
{

  int n = 4096, bx = 32, by = 8;
  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);

  NI = NJ = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;

  double t_start, t_end;

  DATA_TYPE* A;
  DATA_TYPE* B;
  DATA_TYPE* B_outputFromGpu;

  A = (DATA_TYPE*) malloc (NI * NJ * sizeof(DATA_TYPE));
  B = (DATA_TYPE*) malloc (NI * NJ * sizeof(DATA_TYPE));
  B_outputFromGpu = (DATA_TYPE*) malloc (NI * NJ * sizeof(DATA_TYPE));

  //initialize the arrays
  init (A);

  GPU_argv_init ();

#pragma START_TRACING
  convolution2DCuda (A, B, B_outputFromGpu);
#pragma STOP_TRACING
//  t_start = rtclock ();
//  conv2D (A, B);
//  t_end = rtclock ();
//  fprintf (stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start); //);

//  int s = compareResults (B, B_outputFromGpu);
  int s = EXIT_SUCCESS;
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");

  free (A);
  free (B);
  free (B_outputFromGpu);

  return 0;
}

