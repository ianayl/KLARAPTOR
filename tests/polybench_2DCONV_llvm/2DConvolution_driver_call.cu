#include "hip/hip_runtime.h"

#include "2dconv_utils.h"

///////////////////////////////////////
/////// AUTOMATICALLY ANNOTATED ///////
///////////////////////////////////////
#include "kernel_invoker.h"
///////////////////////////////////////
///////////////////////////////////////
const int kernel_info_size_param_idx_Convolution2D_kernel __attribute__((used))  = 2;
const int kernel_info_dim_Convolution2D_kernel __attribute__((used))  = 2;
void
conv2D (DATA_TYPE* A, DATA_TYPE* B)
{
  int i, j;
  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;
  c11 = +0.2;
  c21 = +0.5;
  c31 = -0.8;
  c12 = -0.3;
  c22 = +0.6;
  c32 = -0.9;
  c13 = +0.4;
  c23 = +0.7;
  c33 = +0.10;
  for (i = 1; i < NI - 1; ++i) 
    {
      for (j = 1; j < NJ - 1; ++j) 
 {
   B[i * NJ + j] = c11 * A[(i - 1) * NJ + (j - 1)]
       + c12 * A[(i + 0) * NJ + (j - 1)]
       + c13 * A[(i + 1) * NJ + (j - 1)]
       + c21 * A[(i - 1) * NJ + (j + 0)]
       + c22 * A[(i + 0) * NJ + (j + 0)]
       + c23 * A[(i + 1) * NJ + (j + 0)]
       + c31 * A[(i - 1) * NJ + (j + 1)]
       + c32 * A[(i + 0) * NJ + (j + 1)]
       + c33 * A[(i + 1) * NJ + (j + 1)];
 }
    }
}
void
init (DATA_TYPE* A)
{
  int i, j;
  for (i = 0; i < NI; ++i)
    {
      for (j = 0; j < NJ; ++j)
 {
   A[i * NJ + j] = (float) rand () / RAND_MAX;
 }
    }
}
int
compareResults (DATA_TYPE* B, DATA_TYPE* B_outputFromGpu)
{
  int i, j, fail;
  fail = 0;
  
  for (i = 1; i < (NI - 1); i++)
    {
      for (j = 1; j < (NJ - 1); j++)
 {
   if (percentDiff (
       B[i * NJ + j],       B_outputFromGpu[i * NJ + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
     {
       fail++;
       return (EXIT_FAILURE);
     }
 }
    }
  
  return (EXIT_SUCCESS);
}
void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);
}
//__global__ void
//Convolution2D_kernel (DATA_TYPE *A, DATA_TYPE *B, int NI, int NJ)
//{
//  int j = blockIdx.x * blockDim.x + threadIdx.x;
//  int i = blockIdx.y * blockDim.y + threadIdx.y;
//  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;
//  c11 = +0.2;
//  c21 = +0.5;
//  c31 = -0.8;
//  c12 = -0.3;
//  c22 = +0.6;
//  c32 = -0.9;
//  c13 = +0.4;
//  c23 = +0.7;
//  c33 = +0.10;
//  if ((i < NI - 1) && (j < NJ - 1) && (i > 0) && (j > 0))
//    {
//      B[i * NJ + j] = c11 * A[(i - 1) * NJ + (j - 1)]
//   + c21 * A[(i - 1) * NJ + (j + 0)] + c31 * A[(i - 1) * NJ + (j + 1)]
//   + c12 * A[(i + 0) * NJ + (j - 1)] + c22 * A[(i + 0) * NJ + (j + 0)]
//   + c32 * A[(i + 0) * NJ + (j + 1)] + c13 * A[(i + 1) * NJ + (j - 1)]
//   + c23 * A[(i + 1) * NJ + (j + 0)] + c33 * A[(i + 1) * NJ + (j + 1)];
//    }
//}
void
convolution2DCuda (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* B_outputFromGpu)
{
  cuda_timer t_conv;
  cuda_timer_init (t_conv);
  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;
  hipMalloc ((void **) &A_gpu, sizeof(DATA_TYPE) * NI * NJ);
  hipMalloc ((void **) &B_gpu, sizeof(DATA_TYPE) * NI * NJ);
  hipMemcpy (A_gpu, A, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid ((size_t) ceil (((float) NI) / ((float) block.x)),      (size_t) ceil (((float) NJ) / ((float) block.y)));
  cuda_timer_record_start (t_conv);
  	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_Convolution2D_kernel_0_name[] = "kernel_Convolution2D_kernel_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_Convolution2D_kernel_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_Convolution2D_kernel_sm_75_0_launch_params, grid, block);
 
 void * kernel_Convolution2D_kernel_sm_75_0_kernel_params[]={&A_gpu , &B_gpu , &NI , &NJ};
 
 kernel_invoker(kernel_Convolution2D_kernel_0_name, kernel_Convolution2D_kernel_sm_75_0_launch_params, kernel_Convolution2D_kernel_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t_conv);
  hipDeviceSynchronize ();
  hipMemcpy (B_outputFromGpu, B_gpu, sizeof(DATA_TYPE) * NI * NJ,       hipMemcpyDeviceToHost);
  hipFree (A_gpu);
  hipFree (B_gpu);
  cuda_timer_record_get_elapsed_time (t_conv);
  printf (
      "[trace: n=%d, bx=%d, by=%d, elapsed_Convolution2D_kernel=%0.4f (ms)] ... ",      NI, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t_conv.elapsed_time);
}
int
main (int argc, char **argv)
{
  int n = 4096, bx = 32, by = 8;
  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);
  NI = NJ = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;
  double t_start, t_end;
  DATA_TYPE* A;
  DATA_TYPE* B;
  DATA_TYPE* B_outputFromGpu;
  A = (DATA_TYPE*) malloc (NI * NJ * sizeof(DATA_TYPE));
  B = (DATA_TYPE*) malloc (NI * NJ * sizeof(DATA_TYPE));
  B_outputFromGpu = (DATA_TYPE*) malloc (NI * NJ * sizeof(DATA_TYPE));
  
  init (A);
  GPU_argv_init ();
#pragma START_TRACING
  convolution2DCuda (A, B, B_outputFromGpu);
#pragma STOP_TRACING
  int s = EXIT_SUCCESS;
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");
  free (A);
  free (B);
  free (B_outputFromGpu);
  return 0;
}
