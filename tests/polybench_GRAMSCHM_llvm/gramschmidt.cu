#include "hip/hip_runtime.h"
///**
// * gramschmidt.cu: This file is part of the PolyBench/GPU 1.0 test suite.
// *
// *
// * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
// * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
// * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
// */
//

#include "gramschmidt_utils.h"

#pragma kernel_info_size_param_idx_gramschmidt_kernel1 = 4;
#pragma kernel_info_dim_gramschmidt_kernel1 = 2;

#pragma kernel_info_size_param_idx_gramschmidt_kernel2 = 4;
#pragma kernel_info_dim_gramschmidt_kernel2 = 2;

#pragma kernel_info_size_param_idx_gramschmidt_kernel3 = 4;
#pragma kernel_info_dim_gramschmidt_kernel3 = 2;


void
gramschmidt (DATA_TYPE* A, DATA_TYPE* R, DATA_TYPE* Q)
{
  int i, j, k;
  DATA_TYPE nrm;
  for (k = 0; k < N; k++)
    {
      nrm = 0;
      for (i = 0; i < M; i++)
	{
	  nrm += A[i * N + k] * A[i * N + k];
	}

      R[k * N + k] = sqrt (nrm);
      for (i = 0; i < M; i++)
	{
	  Q[i * N + k] = A[i * N + k] / R[k * N + k];
	}

      for (j = k + 1; j < N; j++)
	{
	  R[k * N + j] = 0;
	  for (i = 0; i < M; i++)
	    {
	      R[k * N + j] += Q[i * N + k] * A[i * N + j];
	    }
	  for (i = 0; i < M; i++)
	    {
	      A[i * N + j] = A[i * N + j] - Q[i * N + k] * R[k * N + j];
	    }
	}
    }
}

void
init_array (DATA_TYPE* A)
{
  int i, j;

  for (i = 0; i < M; i++)
    {
      for (j = 0; j < N; j++)
	{
	  A[i * N + j] = ((DATA_TYPE) (i + 1) * (j + 1)) / (M + 1);
	}
    }
}

int
compareResults (DATA_TYPE* A, DATA_TYPE* A_outputFromGpu)
{
  int i, j, fail;
  fail = 0;

  for (i = 0; i < M; i++)
    {
      for (j = 0; j < N; j++)
	{
	  if (percentDiff (
	      A[i * N + j],
	      A_outputFromGpu[i * N + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
	    {
	      fail++;
	      printf ("i: %d j: %d \n1: %f\n 2: %f\n", i, j, A[i * N + j],
		      A_outputFromGpu[i * N + j]);
	      return (EXIT_FAILURE);
	    }
	}
    }

  return (EXIT_SUCCESS);
  // Print results
//	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
//	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);
  return;
}

__global__ void
gramschmidt_kernel1 (DATA_TYPE *a, DATA_TYPE *r, DATA_TYPE *q, int k, int M,
		     int N)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid == 0)
    {
      DATA_TYPE nrm = 0.0;
      int i;
      for (i = 0; i < M; i++)
	{
	  nrm += a[i * N + k] * a[i * N + k];
	}
      r[k * N + k] = sqrt (nrm);
    }
}

__global__ void
gramschmidt_kernel2 (DATA_TYPE *a, DATA_TYPE *r, DATA_TYPE *q, int k, int M,
		     int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < M)
    {
      q[i * N + k] = a[i * N + k] / r[k * N + k];
    }
}

__global__ void
gramschmidt_kernel3 (DATA_TYPE *a, DATA_TYPE *r, DATA_TYPE *q, int k, int M,
		     int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if ((j > k) && (j < N))
    {
      r[k * N + j] = 0.0;

      int i;
      for (i = 0; i < M; i++)
	{
	  r[k * N + j] += q[i * N + k] * a[i * N + j];
	}

      for (i = 0; i < M; i++)
	{
	  a[i * N + j] -= q[i * N + k] * r[k * N + j];
	}
    }
}

void
gramschmidtCuda (DATA_TYPE* A, DATA_TYPE* R, DATA_TYPE* Q,
		 DATA_TYPE* A_outputFromGpu)
{
  double t_start, t_end;

  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 gridKernel1 (1, 1);
  dim3 gridKernel2 ((size_t) ceil (((float) N) / ((float) DIM_THREAD_BLOCK_X)),
		    1);
  dim3 gridKernel3 ((size_t) ceil (((float) N) / ((float) DIM_THREAD_BLOCK_X)),
		    1);

  DATA_TYPE *A_gpu;
  DATA_TYPE *R_gpu;
  DATA_TYPE *Q_gpu;

  hipMalloc ((void **) &A_gpu, sizeof(DATA_TYPE) * M * N);
  hipMalloc ((void **) &R_gpu, sizeof(DATA_TYPE) * M * N);
  hipMalloc ((void **) &Q_gpu, sizeof(DATA_TYPE) * M * N);
  hipMemcpy (A_gpu, A, sizeof(DATA_TYPE) * M * N, hipMemcpyHostToDevice);

  cuda_timer * t1, *t2, *t3;
  t1 = (cuda_timer*) malloc (N * sizeof(cuda_timer));
  t2 = (cuda_timer*) malloc (N * sizeof(cuda_timer));
  t3 = (cuda_timer*) malloc (N * sizeof(cuda_timer));

  int k;
  for (k = 0; k < N; k++)
    {
      cuda_timer_init (t1[k]);
      cuda_timer_init (t2[k]);
      cuda_timer_init (t3[k]);
    }

//	t_start = rtclock();

  for (k = 0; k < N; k++)
    {
      cuda_timer_record_start (t1[k]);
      gramschmidt_kernel1 <<<gridKernel1, block>>> (A_gpu, R_gpu, Q_gpu, k, M,
						    N);
      cudaCheckKernel()
      ;
      cuda_timer_record_stop (t1[k]);
      hipDeviceSynchronize ();

      cuda_timer_record_start (t2[k]);
      gramschmidt_kernel2 <<<gridKernel2, block>>> (A_gpu, R_gpu, Q_gpu, k, M,
						    N);
      cudaCheckKernel()
      ;
      cuda_timer_record_stop (t2[k]);
      hipDeviceSynchronize ();

      cuda_timer_record_start (t3[k]);
      gramschmidt_kernel3 <<<gridKernel3, block>>> (A_gpu, R_gpu, Q_gpu, k, M,
						    N);
      cudaCheckKernel()
      ;
      cuda_timer_record_stop (t3[k]);
      hipDeviceSynchronize ();
    }
//	t_end = rtclock();
//	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

  float t1_total = 0, t2_total = 0, t3_total = 0;

  for (k = 0; k < N; k++)
    {
      cuda_timer_record_get_elapsed_time (t1[k]);
      cuda_timer_record_get_elapsed_time (t2[k]);
      cuda_timer_record_get_elapsed_time (t3[k]);

      t1_total += t1[k].elapsed_time;
      t2_total += t2[k].elapsed_time;
      t3_total += t3[k].elapsed_time;
    }

  hipMemcpy (A_outputFromGpu, A_gpu, sizeof(DATA_TYPE) * M * N,
	      hipMemcpyDeviceToHost);

  printf (
      "[trace: n=%d, bx=%d, by=%d, elapsed_gramschmidt_kernel1=%0.4f (ms),\n"
      " elapsed_gramschmidt_kernel2=%0.4f (ms), elapsed_gramschmidt_kernel3=%0.4f (ms)] ... ",
      N, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t1_total, t2_total, t3_total);

  hipFree (A_gpu);
  hipFree (R_gpu);
  hipFree (Q_gpu);

  for (k = 0; k < N; k++)
    {
      cuda_timer_destroy (t1[k]);
      cuda_timer_destroy (t2[k]);
      cuda_timer_destroy (t3[k]);
    }
  free (t1);
  free (t2);
  free (t3);

}

int
main (int argc, char *argv[])
{


	int n = 256, bx = 32, by = 8;
  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);

  M=N= n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;

  double t_start, t_end;

  DATA_TYPE* A;
  DATA_TYPE* A_outputFromGpu;
  DATA_TYPE* R;
  DATA_TYPE* Q;

  A = (DATA_TYPE*) malloc (M * N * sizeof(DATA_TYPE));
  A_outputFromGpu = (DATA_TYPE*) malloc (M * N * sizeof(DATA_TYPE));
  R = (DATA_TYPE*) malloc (M * N * sizeof(DATA_TYPE));
  Q = (DATA_TYPE*) malloc (M * N * sizeof(DATA_TYPE));

  init_array (A);

  GPU_argv_init ();
#pragma START_TRACING
  gramschmidtCuda (A, R, Q, A_outputFromGpu);
#pragma STOP_TRACING
//  t_start = rtclock ();
  gramschmidt (A, R, Q);
//  t_end = rtclock ();

//  fprintf (stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

  int s = compareResults (A, A_outputFromGpu);

  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");

  free (A);
  free (A_outputFromGpu);
  free (R);
  free (Q);

  return 0;
}

