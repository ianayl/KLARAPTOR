#include "hip/hip_runtime.h"
///**
// * syr2k.cu: This file is part of the PolyBench/GPU 1.0 test suite.
// *
// *
// * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
// * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
// * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
// */
//

#include "syr2k_utils.h"

#pragma kernel_info_size_param_idx_syr2k_kernel = 3;
#pragma kernel_info_dim_syr2k_kernel = 2;

void
init_arrays (DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
  int i, j;

  for (i = 0; i < N; i++)
    {
      for (j = 0; j < N; j++)
	{
	  C[i * N + j] = ((DATA_TYPE) i * j + 2) / N;
	}

      for (j = 0; j < M; j++)
	{
	  A[i * N + j] = ((DATA_TYPE) i * j) / N;
	  B[i * N + j] = ((DATA_TYPE) i * j + 1) / N;
	}
    }
}

void
syr2k (DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
  int i, j, k;

  for (i = 0; i < N; i++)
    {
      for (j = 0; j < N; j++)
	{
	  C[i * N + j] *= BETA;
	}
    }

  for (i = 0; i < N; i++)
    {
      for (j = 0; j < N; j++)
	{
	  for (k = 0; k < M; k++)
	    {
	      C[i * N + j] += ALPHA * A[i * M + k] * B[j * M + k];
	      C[i * N + j] += ALPHA * B[i * M + k] * A[j * M + k];
	    }
	}
    }
}

int
compareResults (DATA_TYPE *C, DATA_TYPE *C_outputFromGpu)
{
  int i, j, fail;
  fail = 0;

  // Compare C with D
  for (i = 0; i < N; i++)
    {
      for (j = 0; j < N; j++)
	{
	  if (percentDiff (
	      C[i * N + j],
	      C_outputFromGpu[i * N + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
	    {
	      fail++;
	      return (EXIT_FAILURE);
	    }
	}
    }

  return (EXIT_SUCCESS);
  // print results
//	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
//	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);
}

__global__ void
syr2k_kernel (DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c, int M, int N)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < N) && (j < N))
    {
      c[i * N + j] *= BETA;

      int k;
      for (k = 0; k < M; k++)
	{
	  c[i * N + j] += ALPHA * a[i * M + k] * b[j * M + k]
	      + ALPHA * b[i * M + k] * a[j * M + k];
	}
    }
}

void
syr2kCuda (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
  double t_start, t_end;

  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;
  DATA_TYPE *C_gpu;

  hipMalloc ((void **) &A_gpu, sizeof(DATA_TYPE) * N * M);
  hipMalloc ((void **) &B_gpu, sizeof(DATA_TYPE) * N * M);
  hipMalloc ((void **) &C_gpu, sizeof(DATA_TYPE) * N * N);
  hipMemcpy (A_gpu, A, sizeof(DATA_TYPE) * N * M, hipMemcpyHostToDevice);
  hipMemcpy (B_gpu, B, sizeof(DATA_TYPE) * N * M, hipMemcpyHostToDevice);
  hipMemcpy (C_gpu, C, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);

  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid ((size_t) ceil (((float) N) / ((float) DIM_THREAD_BLOCK_X)),
	     (size_t) (ceil (((float) N) / ((float) DIM_THREAD_BLOCK_Y))));

  cuda_timer t;
  cuda_timer_init (t);
  cuda_timer_record_start (t);
//	t_start = rtclock();
  syr2k_kernel <<<grid, block>>> (A_gpu, B_gpu, C_gpu, M, N);

  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t);
  hipDeviceSynchronize ();
//	t_end = rtclock();
//	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

  hipMemcpy (C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * N * N,
	      hipMemcpyDeviceToHost);

  cuda_timer_record_get_elapsed_time (t);
  printf ("[trace: n=%d, bx=%d, by=%d, elapsed_syr2k_kernel=%0.4f (ms)] ... ",
	  N, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t.elapsed_time);
  hipFree (A_gpu);
  hipFree (B_gpu);
  hipFree (C_gpu);
}

int
main (int argc, char ** argv)
{
  int n = 256, bx = 32, by = 8;

  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);

  M = N = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;
  double t_start, t_end;

  DATA_TYPE* A;
  DATA_TYPE* B;
  DATA_TYPE* C;
  DATA_TYPE* C_outputFromGpu;

  A = (DATA_TYPE*) malloc (N * M * sizeof(DATA_TYPE));
  B = (DATA_TYPE*) malloc (N * M * sizeof(DATA_TYPE));
  C = (DATA_TYPE*) malloc (N * M * sizeof(DATA_TYPE));
  C_outputFromGpu = (DATA_TYPE*) malloc (N * M * sizeof(DATA_TYPE));

  init_arrays (A, B, C);

  GPU_argv_init ();
#pragma START_TRACING
	syr2kCuda (A, B, C, C_outputFromGpu);
#pragma STOP_TRACING
//  t_start = rtclock ();
//  syr2k (A, B, C);
//  t_end = rtclock ();
//  fprintf (stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

//  int s = compareResults (C, C_outputFromGpu);
    int s = EXIT_SUCCESS;
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");

  free (A);
  free (B);
  free (C);
  free (C_outputFromGpu);

  return 0;
}

