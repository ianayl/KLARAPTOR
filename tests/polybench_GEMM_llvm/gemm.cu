#include "hip/hip_runtime.h"
///**
// * gemm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
// *
// *
// * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
// * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
// * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
// */

#include "gemm_utils.h"

#pragma kernel_info_size_param_idx_gemm_kernel = 3;
#pragma kernel_info_dim_gemm_kernel = 2;

void
gemm (DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
  int i, j, k;

  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NJ; j++)
	{
	  C[i * NJ + j] *= BETA;

	  for (k = 0; k < NK; ++k)
	    {
	      C[i * NJ + j] += ALPHA * A[i * NK + k] * B[k * NJ + j];
	    }
	}
    }
}

void
init (DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
  int i, j;

  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NK; j++)
	{
	  A[i * NK + j] = ((DATA_TYPE) i * j) / NI;
	}
    }

  for (i = 0; i < NK; i++)
    {
      for (j = 0; j < NJ; j++)
	{
	  B[i * NJ + j] = ((DATA_TYPE) i * j + 1) / NJ;
	}
    }

  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NJ; j++)
	{
	  C[i * NJ + j] = ((DATA_TYPE) i * j + 2) / NJ;
	}
    }
}

int
compareResults (DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
  int i, j, fail;
  fail = 0;

  // Compare C1 and C2
  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NJ; j++)
	{
	  if (percentDiff (
	      C[i * NJ + j],
	      C_outputFromGpu[i * NJ + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
	    {
	      fail++;
	      return (EXIT_FAILURE);
	    }
	}
    }

  return (EXIT_SUCCESS);
  // Print results
//	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
//	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);
}

__global__ void
gemm_kernel (DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c, int NI, int NJ, int NK)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < NI) && (j < NJ))
    {
      c[i * NJ + j] *= BETA;
      int k;
      for (k = 0; k < NK; k++)
	{
	  c[i * NJ + j] += ALPHA * a[i * NK + k] * b[k * NJ + j];
	}
    }
}

void
gemmCuda (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
  double t_start, t_end;

  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;
  DATA_TYPE *C_gpu;

  hipMalloc ((void **) &A_gpu, sizeof(DATA_TYPE) * NI * NK);
  hipMalloc ((void **) &B_gpu, sizeof(DATA_TYPE) * NK * NJ);
  hipMalloc ((void **) &C_gpu, sizeof(DATA_TYPE) * NI * NJ);

  hipMemcpy (A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
  hipMemcpy (B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
  hipMemcpy (C_gpu, C, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);

  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid ((size_t) (ceil (((float) NI) / ((float) block.x))),
	     (size_t) (ceil (((float) NJ) / ((float) block.y))));

//	t_start = rtclock();
  cuda_timer t;
  cuda_timer_init (t);
  cuda_timer_record_start (t);

  gemm_kernel <<<grid, block>>> (A_gpu, B_gpu, C_gpu, NI, NJ, NK);

  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t);
  hipDeviceSynchronize ();

  cuda_timer_record_get_elapsed_time (t);
//	t_end = rtclock();
//	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

  hipMemcpy (C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI * NJ,
	      hipMemcpyDeviceToHost);

  printf ("[trace: n=%d, bx=%d, by=%d, elapsed_gemm_kernel=%0.4f (ms)] ... ",
	  NI, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t.elapsed_time);
  hipFree (A_gpu);
  hipFree (B_gpu);
  hipFree (C_gpu);
}

int
main (int argc, char *argv[])
{
//	double t_start, t_end;

  int n = 256, bx = 32, by = 8;

  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);

  NI = NJ = NK = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;

  DATA_TYPE* A;
  DATA_TYPE* B;
  DATA_TYPE* C;
  DATA_TYPE* C_outputFromGpu;

  A = (DATA_TYPE*) malloc (NI * NK * sizeof(DATA_TYPE));
  B = (DATA_TYPE*) malloc (NK * NJ * sizeof(DATA_TYPE));
  C = (DATA_TYPE*) malloc (NI * NJ * sizeof(DATA_TYPE));
  C_outputFromGpu = (DATA_TYPE*) malloc (NI * NJ * sizeof(DATA_TYPE));

  init (A, B, C);

  GPU_argv_init ();

#pragma START_TRACING
  gemmCuda (A, B, C, C_outputFromGpu);
#pragma STOP_TRACING 

//	t_start = rtclock();
  gemm (A, B, C);
//	t_end = rtclock();
//	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

  int s = compareResults (C, C_outputFromGpu);
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");

  free (A);
  free (B);
  free (C);
  free (C_outputFromGpu);

  return 0;
}

