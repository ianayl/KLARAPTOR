#include "hip/hip_runtime.h"
///**
// * syrk.cu: This file is part of the PolyBench/GPU 1.0 test suite.
// *
// *
// * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
// * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
// * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
// */
//

#include "syrk_utils.h"

#pragma kernel_info_size_param_idx_syrk_kernel = 4;
#pragma kernel_info_dim_syrk_kernel = 2;


void
init_arrays (DATA_TYPE* A, DATA_TYPE* C)
{
  int i, j;

  for (i = 0; i < N; i++)
    {
      for (j = 0; j < M; j++)
	{
	  A[i * M + j] = ((DATA_TYPE) i * j) / N;
	}

      for (j = 0; j < N; j++)
	{
	  C[i * M + j] = ((DATA_TYPE) i * j + 2) / N;
	}
    }
}

void
syrk (DATA_TYPE* A, DATA_TYPE* C)
{
  int i, j, k;

  /*  C := alpha*A*A' + beta*C */
  for (i = 0; i < N; i++)
    {
      for (j = 0; j < N; j++)
	{
	  C[i * M + j] *= beta;
	}
    }

  for (i = 0; i < N; i++)
    {
      for (j = 0; j < N; j++)
	{
	  for (k = 0; k < M; k++)
	    {
	      C[i * N + j] += alpha * A[i * M + k] * A[j * M + k];
	    }
	}
    }
}

int
compareResults (DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
  int i, j, fail;
  fail = 0;

  // Compare C with D
  for (i = 0; i < N; i++)
    {
      for (j = 0; j < M; j++)
	{
	  if (percentDiff (
	      C[i * M + j],
	      C_outputFromGpu[i * M + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
	    {
	      fail++;
	      return (EXIT_FAILURE);
	    }
	}
    }

  return (EXIT_SUCCESS);
  // print results
//	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
//	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);

  return;
}

__global__ void
syrk_kernel (DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE *a, DATA_TYPE *c, int M,
	     int N)
{
  /*  C := alpha*A*A' + beta*C */
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < N) && (j < N))
    {
      c[i * N + j] *= beta;
      int k;
      for (k = 0; k < M; k++)
	{
	  c[i * N + j] += alpha * a[i * M + k] * a[j * M + k];
	}
    }
}

void
syrkCuda (DATA_TYPE* A, DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
  double t_start, t_end;

  DATA_TYPE* A_gpu;
  DATA_TYPE* C_gpu;

  hipMalloc ((void **) &A_gpu, sizeof(DATA_TYPE) * N * M);
  hipMalloc ((void **) &C_gpu, sizeof(DATA_TYPE) * N * N);
  hipMemcpy (A_gpu, A, sizeof(DATA_TYPE) * N * M, hipMemcpyHostToDevice);
  hipMemcpy (C_gpu, C, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);

  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid ((size_t) (ceil (((float) N) / ((float) DIM_THREAD_BLOCK_X))),
	     (size_t) ceil (((float) N) / ((float) DIM_THREAD_BLOCK_Y)));

  cuda_timer t;
  cuda_timer_init (t);

//	t_start = rtclock();
  cuda_timer_record_start (t);
  syrk_kernel <<<grid, block>>> (alpha, beta, A_gpu, C_gpu, M, N);
  cuda_timer_record_stop (t);
  cudaCheckKernel()
  ;
  hipDeviceSynchronize ();
//	t_end = rtclock();
//	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

  hipMemcpy (C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * N * N,
	      hipMemcpyDeviceToHost);
  cuda_timer_record_get_elapsed_time (t);
  printf ("[trace: n=%d, bx=%d, by=%d, elapsed_syrk_kernel=%0.4f (ms)] ... ", N,
	  DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t.elapsed_time);
  hipFree (A_gpu);
  hipFree (C_gpu);
}

int
main (int argc, char ** argv)
{
  int n = 256, bx = 32, by = 8;

  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);

  M = N = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;

  DATA_TYPE* A;
  DATA_TYPE* C;
  DATA_TYPE* C_outputFromGpu;

  A = (DATA_TYPE*) malloc (N * M * sizeof(DATA_TYPE));
  C = (DATA_TYPE*) malloc (N * M * sizeof(DATA_TYPE));
  C_outputFromGpu = (DATA_TYPE*) malloc (N * M * sizeof(DATA_TYPE));

  init_arrays (A, C);

  GPU_argv_init ();

#pragma START_TRACING
  syrkCuda (A, C, C_outputFromGpu);
#pragma STOP_TRACING

//  t_start = rtclock ();
  syrk (A, C);
//  t_end = rtclock ();
//  fprintf (stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

  int s = compareResults (C, C_outputFromGpu);
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");

  free (A);
  free (C);
  free (C_outputFromGpu);

  return 0;
}

