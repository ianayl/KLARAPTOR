#include "hip/hip_runtime.h"
///**
// * 3DConvolution.cu: This file is part of the PolyBench/GPU 1.0 test suite.
// *
// *
// * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
// * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
// * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
// */
//

#include "3dconv_utils.h"
#pragma kernel_info_size_param_idx_convolution3D_kernel = 3;
#pragma kernel_info_dim_convolution3D_kernel = 2;

void
conv3D (DATA_TYPE* A, DATA_TYPE* B)
{
  int i, j, k;
  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

  c11 = +2;
  c21 = +5;
  c31 = -8;
  c12 = -3;
  c22 = +6;
  c32 = -9;
  c13 = +4;
  c23 = +7;
  c33 = +10;

  for (i = 1; i < NI - 1; ++i) // 0
    {
      for (j = 1; j < NJ - 1; ++j) // 1
	{
	  for (k = 1; k < NK - 1; ++k) // 2
	    {
	      //printf("i:%d\nj:%d\nk:%d\n", i, j, k);
	      B[i * (NK * NJ) + j * NK + k] = c11
		  * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
		  + c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
		  + c21 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
		  + c23 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
		  + c31 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
		  + c33 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
		  + c12 * A[(i + 0) * (NK * NJ) + (j - 1) * NK + (k + 0)]
		  + c22 * A[(i + 0) * (NK * NJ) + (j + 0) * NK + (k + 0)]
		  + c32 * A[(i + 0) * (NK * NJ) + (j + 1) * NK + (k + 0)]
		  + c11 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k + 1)]
		  + c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k + 1)]
		  + c21 * A[(i - 1) * (NK * NJ) + (j + 0) * NK + (k + 1)]
		  + c23 * A[(i + 1) * (NK * NJ) + (j + 0) * NK + (k + 1)]
		  + c31 * A[(i - 1) * (NK * NJ) + (j + 1) * NK + (k + 1)]
		  + c33 * A[(i + 1) * (NK * NJ) + (j + 1) * NK + (k + 1)];
	    }
	}
    }
}

void
init (DATA_TYPE* A)
{
  int i, j, k;

  for (i = 0; i < NI; ++i)
    {
      for (j = 0; j < NJ; ++j)
	{
	  for (k = 0; k < NK; ++k)
	    {
	      A[i * (NK * NJ) + j * NK + k] = i % 12 + 2 * (j % 7)
		  + 3 * (k % 13);
	    }
	}
    }
}

int
compareResults (DATA_TYPE* B, DATA_TYPE* B_outputFromGpu)
{
  int i, j, k, fail;
  fail = 0;

  // Compare result from cpu and gpu...
  for (i = 1; i < NI - 1; ++i) // 0
    {
      for (j = 1; j < NJ - 1; ++j) // 1
	{
	  for (k = 1; k < NK - 1; ++k) // 2
	    {
	      if (percentDiff (
		  B[i * (NK * NJ) + j * NK + k],
		  B_outputFromGpu[i * (NK * NJ) + j * NK + k]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
		  fail++;
		  return (EXIT_FAILURE);

		}
	    }
	}
    }

  // Print results
//  printf (
//      "Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n",
//      PERCENT_DIFF_ERROR_THRESHOLD,
//      fail);
  return (EXIT_SUCCESS);
}

void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
//  printf ("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);
}

__global__ void
convolution3D_kernel (DATA_TYPE *A, DATA_TYPE *B, int i, int NI, int NJ, int NK)
{
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

  c11 = +2;
  c21 = +5;
  c31 = -8;
  c12 = -3;
  c22 = +6;
  c32 = -9;
  c13 = +4;
  c23 = +7;
  c33 = +10;

  if ((i < (NI - 1)) && (j < (NJ - 1)) && (k < (NK - 1)) && (i > 0) && (j > 0)
      && (k > 0))
    {
      B[i * (NK * NJ) + j * NK + k] = c11
	  * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
	  + c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
	  + c21 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
	  + c23 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
	  + c31 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
	  + c33 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
	  + c12 * A[(i + 0) * (NK * NJ) + (j - 1) * NK + (k + 0)]
	  + c22 * A[(i + 0) * (NK * NJ) + (j + 0) * NK + (k + 0)]
	  + c32 * A[(i + 0) * (NK * NJ) + (j + 1) * NK + (k + 0)]
	  + c11 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k + 1)]
	  + c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k + 1)]
	  + c21 * A[(i - 1) * (NK * NJ) + (j + 0) * NK + (k + 1)]
	  + c23 * A[(i + 1) * (NK * NJ) + (j + 0) * NK + (k + 1)]
	  + c31 * A[(i - 1) * (NK * NJ) + (j + 1) * NK + (k + 1)]
	  + c33 * A[(i + 1) * (NK * NJ) + (j + 1) * NK + (k + 1)];
    }
}

void
convolution3DCuda (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* B_outputFromGpu)
{
  double t_start, t_end;

  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;

  hipMalloc ((void **) &A_gpu, sizeof(DATA_TYPE) * NI * NJ * NK);
  hipMalloc ((void **) &B_gpu, sizeof(DATA_TYPE) * NI * NJ * NK);
  hipMemcpy (A_gpu, A, sizeof(DATA_TYPE) * NI * NJ * NK,
	      hipMemcpyHostToDevice);
  hipMemcpy (B_gpu, B, sizeof(DATA_TYPE) * NI * NJ * NK,
	      hipMemcpyHostToDevice);

  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid ((size_t) (ceil (((float) NK) / ((float) block.x))),
	     (size_t) (ceil (((float) NJ) / ((float) block.y))));

//  t_start = rtclock ();
  cuda_timer t_conv3d;
  cuda_timer_init (t_conv3d);

  int i;
  cuda_timer_record_start (t_conv3d);
  for (i = 1; i < NI - 1; ++i) // 0
    {
      convolution3D_kernel <<<grid, block>>> (A_gpu, B_gpu, i, NI, NJ, NK);
      cudaCheckKernel()
      ;
    }
  cuda_timer_record_stop (t_conv3d);

  hipDeviceSynchronize ();
//  t_end = rtclock ();
//  fprintf (stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

  cuda_timer_record_get_elapsed_time (t_conv3d);
  hipMemcpy (B_outputFromGpu, B_gpu, sizeof(DATA_TYPE) * NI * NJ * NK,
	      hipMemcpyDeviceToHost);

  printf (
      "[trace: n=%d, bx=%d, by=%d, elapsed_convolution3D_kernel=%0.4f (ms)] ... ",
      NI, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t_conv3d.elapsed_time);
  hipFree (A_gpu);
  hipFree (B_gpu);

}

int
main (int argc, char *argv[])
{
  int n = 256, bx = 32, by = 8;

  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);

  NI = NJ = NK = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;

//  double t_start, t_end;

  DATA_TYPE* A;
  DATA_TYPE* B;
  DATA_TYPE* B_outputFromGpu;

  A = (DATA_TYPE*) malloc (NI * NJ * NK * sizeof(DATA_TYPE));
  B = (DATA_TYPE*) malloc (NI * NJ * NK * sizeof(DATA_TYPE));
  B_outputFromGpu = (DATA_TYPE*) malloc (NI * NJ * NK * sizeof(DATA_TYPE));

  init (A);

  GPU_argv_init ();

#pragma START_TRACING
  convolution3DCuda (A, B, B_outputFromGpu);
#pragma STOP_TRACING

//  t_start = rtclock ();
  conv3D (A, B);
//  t_end = rtclock ();
//  fprintf (stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

  int s = compareResults (B, B_outputFromGpu);
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");

  free (A);
  free (B);
  free (B_outputFromGpu);

  return 0;
}

