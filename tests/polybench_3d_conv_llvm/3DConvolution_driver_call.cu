#include "hip/hip_runtime.h"

#include "3dconv_utils.h"

///////////////////////////////////////
/////// AUTOMATICALLY ANNOTATED ///////
///////////////////////////////////////
#include "kernel_invoker.h"
///////////////////////////////////////
///////////////////////////////////////
const int kernel_info_size_param_idx_convolution3D_kernel __attribute__((used))  = 3;
const int kernel_info_dim_convolution3D_kernel __attribute__((used))  = 2;
void
conv3D (DATA_TYPE* A, DATA_TYPE* B)
{
  int i, j, k;
  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;
  c11 = +2;
  c21 = +5;
  c31 = -8;
  c12 = -3;
  c22 = +6;
  c32 = -9;
  c13 = +4;
  c23 = +7;
  c33 = +10;
  for (i = 1; i < NI - 1; ++i) 
    {
      for (j = 1; j < NJ - 1; ++j) 
 {
   for (k = 1; k < NK - 1; ++k) 
     {
       
       B[i * (NK * NJ) + j * NK + k] = c11
    * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
    + c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
    + c21 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
    + c23 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
    + c31 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
    + c33 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
    + c12 * A[(i + 0) * (NK * NJ) + (j - 1) * NK + (k + 0)]
    + c22 * A[(i + 0) * (NK * NJ) + (j + 0) * NK + (k + 0)]
    + c32 * A[(i + 0) * (NK * NJ) + (j + 1) * NK + (k + 0)]
    + c11 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k + 1)]
    + c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k + 1)]
    + c21 * A[(i - 1) * (NK * NJ) + (j + 0) * NK + (k + 1)]
    + c23 * A[(i + 1) * (NK * NJ) + (j + 0) * NK + (k + 1)]
    + c31 * A[(i - 1) * (NK * NJ) + (j + 1) * NK + (k + 1)]
    + c33 * A[(i + 1) * (NK * NJ) + (j + 1) * NK + (k + 1)];
     }
 }
    }
}
void
init (DATA_TYPE* A)
{
  int i, j, k;
  for (i = 0; i < NI; ++i)
    {
      for (j = 0; j < NJ; ++j)
 {
   for (k = 0; k < NK; ++k)
     {
       A[i * (NK * NJ) + j * NK + k] = i % 12 + 2 * (j % 7)
    + 3 * (k % 13);
     }
 }
    }
}
int
compareResults (DATA_TYPE* B, DATA_TYPE* B_outputFromGpu)
{
  int i, j, k, fail;
  fail = 0;
  
  for (i = 1; i < NI - 1; ++i) 
    {
      for (j = 1; j < NJ - 1; ++j) 
 {
   for (k = 1; k < NK - 1; ++k) 
     {
       if (percentDiff (
    B[i * (NK * NJ) + j * NK + k],    B_outputFromGpu[i * (NK * NJ) + j * NK + k]) > PERCENT_DIFF_ERROR_THRESHOLD)
  {
    fail++;
    return (EXIT_FAILURE);
  }
     }
 }
    }
  
  return (EXIT_SUCCESS);
}
void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);
}
//__global__ void
//convolution3D_kernel (DATA_TYPE *A, DATA_TYPE *B, int i, int NI, int NJ, int NK)
//{
//  int k = blockIdx.x * blockDim.x + threadIdx.x;
//  int j = blockIdx.y * blockDim.y + threadIdx.y;
//  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;
//  c11 = +2;
//  c21 = +5;
//  c31 = -8;
//  c12 = -3;
//  c22 = +6;
//  c32 = -9;
//  c13 = +4;
//  c23 = +7;
//  c33 = +10;
//  if ((i < (NI - 1)) && (j < (NJ - 1)) && (k < (NK - 1)) && (i > 0) && (j > 0)
//      && (k > 0))
//    {
//      B[i * (NK * NJ) + j * NK + k] = c11
//   * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
//   + c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
//   + c21 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
//   + c23 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
//   + c31 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
//   + c33 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)]
//   + c12 * A[(i + 0) * (NK * NJ) + (j - 1) * NK + (k + 0)]
//   + c22 * A[(i + 0) * (NK * NJ) + (j + 0) * NK + (k + 0)]
//   + c32 * A[(i + 0) * (NK * NJ) + (j + 1) * NK + (k + 0)]
//   + c11 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k + 1)]
//   + c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k + 1)]
//   + c21 * A[(i - 1) * (NK * NJ) + (j + 0) * NK + (k + 1)]
//   + c23 * A[(i + 1) * (NK * NJ) + (j + 0) * NK + (k + 1)]
//   + c31 * A[(i - 1) * (NK * NJ) + (j + 1) * NK + (k + 1)]
//   + c33 * A[(i + 1) * (NK * NJ) + (j + 1) * NK + (k + 1)];
//    }
//}
void
convolution3DCuda (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* B_outputFromGpu)
{
  double t_start, t_end;
  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;
  hipMalloc ((void **) &A_gpu, sizeof(DATA_TYPE) * NI * NJ * NK);
  hipMalloc ((void **) &B_gpu, sizeof(DATA_TYPE) * NI * NJ * NK);
  hipMemcpy (A_gpu, A, sizeof(DATA_TYPE) * NI * NJ * NK,       hipMemcpyHostToDevice);
  hipMemcpy (B_gpu, B, sizeof(DATA_TYPE) * NI * NJ * NK,       hipMemcpyHostToDevice);
  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid ((size_t) (ceil (((float) NK) / ((float) block.x))),      (size_t) (ceil (((float) NJ) / ((float) block.y))));
  cuda_timer t_conv3d;
  cuda_timer_init (t_conv3d);
  int i;
  cuda_timer_record_start (t_conv3d);
  for (i = 1; i < NI - 1; ++i) 
    {
      	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_convolution3D_kernel_0_name[] = "kernel_convolution3D_kernel_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_convolution3D_kernel_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_convolution3D_kernel_sm_75_0_launch_params, grid, block);
 
 void * kernel_convolution3D_kernel_sm_75_0_kernel_params[]={&A_gpu , &B_gpu , &i , &NI , &NJ , &NK};
 
 kernel_invoker(kernel_convolution3D_kernel_0_name, kernel_convolution3D_kernel_sm_75_0_launch_params, kernel_convolution3D_kernel_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
      cudaCheckKernel()
      ;
    }
  cuda_timer_record_stop (t_conv3d);
  hipDeviceSynchronize ();
  cuda_timer_record_get_elapsed_time (t_conv3d);
  hipMemcpy (B_outputFromGpu, B_gpu, sizeof(DATA_TYPE) * NI * NJ * NK,       hipMemcpyDeviceToHost);
  printf (
      "[trace: n=%d, bx=%d, by=%d, elapsed_convolution3D_kernel=%0.4f (ms)] ... ",      NI, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t_conv3d.elapsed_time);
  hipFree (A_gpu);
  hipFree (B_gpu);
}
int
main (int argc, char *argv[])
{
  int n = 256, bx = 32, by = 8;
  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);
  NI = NJ = NK = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;
  DATA_TYPE* A;
  DATA_TYPE* B;
  DATA_TYPE* B_outputFromGpu;
  A = (DATA_TYPE*) malloc (NI * NJ * NK * sizeof(DATA_TYPE));
  B = (DATA_TYPE*) malloc (NI * NJ * NK * sizeof(DATA_TYPE));
  B_outputFromGpu = (DATA_TYPE*) malloc (NI * NJ * NK * sizeof(DATA_TYPE));
  init (A);
  GPU_argv_init ();
#pragma START_TRACING
  convolution3DCuda (A, B, B_outputFromGpu);
#pragma STOP_TRACING
  conv3D (A, B);
  int s = compareResults (B, B_outputFromGpu);
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");
  free (A);
  free (B);
  free (B_outputFromGpu);
  return 0;
}
