#include "hip/hip_runtime.h"

#include "fdtd2d_utils.h"

///////////////////////////////////////
/////// AUTOMATICALLY ANNOTATED ///////
///////////////////////////////////////
#include "kernel_invoker.h"
///////////////////////////////////////
///////////////////////////////////////
const int kernel_info_size_param_idx_fdtd_step1_kernel __attribute__((used))  = 5;
const int kernel_info_dim_fdtd_step1_kernel __attribute__((used))  = 2;
const int kernel_info_size_param_idx_fdtd_step2_kernel __attribute__((used))  = 4;
const int kernel_info_dim_fdtd_step2_kernel __attribute__((used))  = 2;
const int kernel_info_size_param_idx_fdtd_step3_kernel __attribute__((used))  = 4;
const int kernel_info_dim_fdtd_step3_kernel __attribute__((used))  = 2;
void
init_arrays (DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz)
{
  int i, j;
  for (i = 0; i < tmax; i++)
    {
      _fict_[i] = (DATA_TYPE) i;
    }
  for (i = 0; i < NX; i++)
    {
      for (j = 0; j < NY; j++)
 {
   ex[i * NY + j] = ((DATA_TYPE) i * (j + 1) + 1) / NX;
   ey[i * NY + j] = ((DATA_TYPE) (i - 1) * (j + 2) + 2) / NX;
   hz[i * NY + j] = ((DATA_TYPE) (i - 9) * (j + 4) + 3) / NX;
 }
    }
}
void
runFdtd (DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz)
{
  int t, i, j;
  for (t = 0; t < tmax; t++)
    {
      for (j = 0; j < NY; j++)
 {
   ey[0 * NY + j] = _fict_[t];
 }
      for (i = 1; i < NX; i++)
 {
   for (j = 0; j < NY; j++)
     {
       ey[i * NY + j] = ey[i * NY + j]
    - 0.5 * (hz[i * NY + j] - hz[(i - 1) * NY + j]);
     }
 }
      for (i = 0; i < NX; i++)
 {
   for (j = 1; j < NY; j++)
     {
       ex[i * (NY + 1) + j] = ex[i * (NY + 1) + j]
    - 0.5 * (hz[i * NY + j] - hz[i * NY + (j - 1)]);
     }
 }
      for (i = 0; i < NX; i++)
 {
   for (j = 0; j < NY; j++)
     {
       hz[i * NY + j] = hz[i * NY + j]
    - 0.7
        * (ex[i * (NY + 1) + (j + 1)] - ex[i * (NY + 1) + j]
     + ey[(i + 1) * NY + j] - ey[i * NY + j]);
     }
 }
    }
}
int
compareResults (DATA_TYPE* hz1, DATA_TYPE* hz2)
{
  int i, j, fail;
  fail = 0;
  for (i = 0; i < NX; i++)
    {
      for (j = 0; j < NY; j++)
 {
   if (percentDiff (hz1[i * NY + j],      hz2[i * NY + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
     {
       fail++;
       return (EXIT_FAILURE);
     }
 }
    }
  
  return (EXIT_SUCCESS);
}
void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);
}
//__global__ void
//fdtd_step1_kernel (DATA_TYPE* _fict_, DATA_TYPE *ex, DATA_TYPE *ey,     DATA_TYPE *hz, int t, int NX, int NY)
//{
//  int j = blockIdx.x * blockDim.x + threadIdx.x;
//  int i = blockIdx.y * blockDim.y + threadIdx.y;
//  if ((i < NX) && (j < NY))
//    {
//      if (i == 0)
// {
//   ey[i * NY + j] = _fict_[t];
// }
//      else
// {
//   ey[i * NY + j] = ey[i * NY + j]
//       - 0.5f * (hz[i * NY + j] - hz[(i - 1) * NY + j]);
// }
//    }
//}
//__global__ void
//fdtd_step2_kernel (DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t, int NX,     int NY)
//{
//  int j = blockIdx.x * blockDim.x + threadIdx.x;
//  int i = blockIdx.y * blockDim.y + threadIdx.y;
//  if ((i < NX) && (j < NY) && (j > 0))
//    {
//      ex[i * (NY + 1) + j] = ex[i * (NY + 1) + j]
//   - 0.5f * (hz[i * NY + j] - hz[i * NY + (j - 1)]);
//    }
//}
//__global__ void
//fdtd_step3_kernel (DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t, int NX,     int NY)
//{
//  int j = blockIdx.x * blockDim.x + threadIdx.x;
//  int i = blockIdx.y * blockDim.y + threadIdx.y;
//  if ((i < NX) && (j < NY))
//    {
//      hz[i * NY + j] = hz[i * NY + j]
//   - 0.7f
//       * (ex[i * (NY + 1) + (j + 1)] - ex[i * (NY + 1) + j]
//    + ey[(i + 1) * NY + j] - ey[i * NY + j]);
//    }
//}
void
fdtdCuda (DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz,   DATA_TYPE* hz_outputFromGpu)
{
  DATA_TYPE *_fict_gpu;
  DATA_TYPE *ex_gpu;
  DATA_TYPE *ey_gpu;
  DATA_TYPE *hz_gpu;
  hipMalloc ((void **) &_fict_gpu, sizeof(DATA_TYPE) * tmax);
  hipMalloc ((void **) &ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1));
  hipMalloc ((void **) &ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY);
  hipMalloc ((void **) &hz_gpu, sizeof(DATA_TYPE) * NX * NY);
  hipMemcpy (_fict_gpu, _fict_, sizeof(DATA_TYPE) * tmax,       hipMemcpyHostToDevice);
  hipMemcpy (ex_gpu, ex, sizeof(DATA_TYPE) * NX * (NY + 1),       hipMemcpyHostToDevice);
  hipMemcpy (ey_gpu, ey, sizeof(DATA_TYPE) * (NX + 1) * NY,       hipMemcpyHostToDevice);
  hipMemcpy (hz_gpu, hz, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid ((size_t) ceil (((float) NY) / ((float) block.x)),      (size_t) ceil (((float) NX) / ((float) block.y)));
  cuda_timer * t1, *t2, *t3;
  t1 = (cuda_timer*) malloc (tmax * sizeof(cuda_timer));
  t2 = (cuda_timer*) malloc (tmax * sizeof(cuda_timer));
  t3 = (cuda_timer*) malloc (tmax * sizeof(cuda_timer));
  for (int t = 0; t < tmax; t++)
    {
      cuda_timer_init (t1[t]);
      cuda_timer_init (t2[t]);
      cuda_timer_init (t3[t]);
    }
  for (int t = 0; t < tmax; t++)
    {
      cuda_timer_record_start (t1[t]);
      	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_fdtd_step1_kernel_0_name[] = "kernel_fdtd_step1_kernel_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_fdtd_step1_kernel_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_fdtd_step1_kernel_sm_75_0_launch_params, grid, block);
 
 void * kernel_fdtd_step1_kernel_sm_75_0_kernel_params[]={&_fict_gpu , &ex_gpu , &ey_gpu , &hz_gpu , &t , &NX , &NY};
 
 kernel_invoker(kernel_fdtd_step1_kernel_0_name, kernel_fdtd_step1_kernel_sm_75_0_launch_params, kernel_fdtd_step1_kernel_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
      cudaCheckKernel()
      ;
      cuda_timer_record_stop (t1[t]);
      hipDeviceSynchronize ();
      cuda_timer_record_start (t2[t]);
      	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_fdtd_step2_kernel_0_name[] = "kernel_fdtd_step2_kernel_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_fdtd_step2_kernel_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_fdtd_step2_kernel_sm_75_0_launch_params, grid, block);
 
 void * kernel_fdtd_step2_kernel_sm_75_0_kernel_params[]={&ex_gpu , &ey_gpu , &hz_gpu , &t , &NX , &NY};
 
 kernel_invoker(kernel_fdtd_step2_kernel_0_name, kernel_fdtd_step2_kernel_sm_75_0_launch_params, kernel_fdtd_step2_kernel_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
      cudaCheckKernel()
      ;
      cuda_timer_record_stop (t2[t]);
      hipDeviceSynchronize ();
      cuda_timer_record_start (t3[t]);
      	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_fdtd_step3_kernel_0_name[] = "kernel_fdtd_step3_kernel_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_fdtd_step3_kernel_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_fdtd_step3_kernel_sm_75_0_launch_params, grid, block);
 
 void * kernel_fdtd_step3_kernel_sm_75_0_kernel_params[]={&ex_gpu , &ey_gpu , &hz_gpu , &t , &NX , &NY};
 
 kernel_invoker(kernel_fdtd_step3_kernel_0_name, kernel_fdtd_step3_kernel_sm_75_0_launch_params, kernel_fdtd_step3_kernel_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
      cudaCheckKernel()
      ;
      cuda_timer_record_stop (t3[t]);
      hipDeviceSynchronize ();
    }
  hipMemcpy (hz_outputFromGpu, hz_gpu, sizeof(DATA_TYPE) * NX * NY,       hipMemcpyDeviceToHost);
  float t1_total = 0, t2_total = 0, t3_total = 0;
  for (int t = 0; t < tmax; t++)
    {
      cuda_timer_record_get_elapsed_time (t1[t]);
      cuda_timer_record_get_elapsed_time (t2[t]);
      cuda_timer_record_get_elapsed_time (t3[t]);
      t1_total += t1[t].elapsed_time;
      t2_total += t2[t].elapsed_time;
      t3_total += t3[t].elapsed_time;
    }
  printf (
        "[trace: n=%d, bx=%d, by=%d, elapsed_fdtd_step1_kernel=%0.4f (ms),\n "
        "elapsed_fdtd_step2_kernel=%0.4f (ms), elapsed_fdtd_step3_kernel=%0.4f (ms)] ... ",        NX, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t1_total,        t2_total,        t3_total
        );
  hipFree (_fict_gpu);
  hipFree (ex_gpu);
  hipFree (ey_gpu);
  hipFree (hz_gpu);
  for (int t = 0; t < tmax; t++)
    {
      cuda_timer_destroy (t1[t]);
      cuda_timer_destroy (t2[t]);
      cuda_timer_destroy (t3[t]);
    }
}
int
main (int argc, char ** argv)
{
  int n = 256, bx = 32, by = 8;
  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);
  NX = NY = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;
  DATA_TYPE* _fict_;
  DATA_TYPE* ex;
  DATA_TYPE* ey;
  DATA_TYPE* hz;
  DATA_TYPE* hz_outputFromGpu;
  _fict_ = (DATA_TYPE*) malloc (tmax * sizeof(DATA_TYPE));
  ex = (DATA_TYPE*) malloc (NX * (NY + 1) * sizeof(DATA_TYPE));
  ey = (DATA_TYPE*) malloc ((NX + 1) * NY * sizeof(DATA_TYPE));
  hz = (DATA_TYPE*) malloc (NX * NY * sizeof(DATA_TYPE));
  hz_outputFromGpu = (DATA_TYPE*) malloc (NX * NY * sizeof(DATA_TYPE));
  init_arrays (_fict_, ex, ey, hz);
  GPU_argv_init ();
#pragma START_TRACING
  fdtdCuda (_fict_, ex, ey, hz, hz_outputFromGpu);
#pragma STOP_TRACING
  runFdtd (_fict_, ex, ey, hz);
  int s = compareResults (hz, hz_outputFromGpu);
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");
  free (_fict_);
  free (ex);
  free (ey);
  free (hz);
  free (hz_outputFromGpu);
  return 0;
}
