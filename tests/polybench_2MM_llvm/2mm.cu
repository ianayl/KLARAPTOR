#include "hip/hip_runtime.h"
///**
// * 2mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
// *
// *
// * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
// * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
// * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
// */
//
#include "2mm_utils.h"

#pragma kernel_info_size_param_idx_mm2_kernel1 = 3;
#pragma kernel_info_size_param_idx_mm2_kernel2 = 3;
#pragma kernel_info_dim_mm2_kernel1 = 2;
#pragma kernel_info_dim_mm2_kernel2 = 2;

void
init_array (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D)
{
  int i, j;

  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NK; j++)
	{
	  A[i * NI + j] = ((DATA_TYPE) i * j) / NI;
	}
    }

  for (i = 0; i < NK; i++)
    {
      for (j = 0; j < NJ; j++)
	{
	  B[i * NK + j] = ((DATA_TYPE) i * (j + 1)) / NJ;
	}
    }

  for (i = 0; i < NL; i++)
    {
      for (j = 0; j < NJ; j++)
	{
	  C[i * NL + j] = ((DATA_TYPE) i * (j + 3)) / NL;
	}
    }

  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NL; j++)
	{
	  D[i * NL + j] = ((DATA_TYPE) i * (j + 2)) / NK;
	}
    }
}

int
compareResults (DATA_TYPE *E, DATA_TYPE *E_outputFromGpu)
{
  int i, j, fail;
  fail = 0;

  for (i = 0; i < NL; i++)
    {
      for (j = 0; j < NI; j++)
	{
	  if (percentDiff (
	      E[i * NI + j],
	      E_outputFromGpu[i * NI + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
	    {
	      fail++;
	      return (EXIT_FAILURE);
	    }
	}
    }

  return EXIT_SUCCESS;
  // print results
//	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
//  printf ("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);
}

__global__ void
mm2_kernel1 (DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C, int NI, int NJ, int NK,
	     int NL)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < NI) && (j < NJ))
    {
      int k;
      for (k = 0; k < NK; k++)
	{
	  C[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
	}
    }
}

__global__ void
mm2_kernel2 (DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *E, int NI, int NJ, int NK,
	     int NL)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < NI) && (j < NL))
    {
      int k;
      for (k = 0; k < NJ; k++)
	{
	  E[i * NL + j] += C[i * NJ + k] * D[k * NL + j];
	}
    }
}

void
mm2_cpu (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E)
{
  int i, j, k;

  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NJ; j++)
	{
	  C[i * NJ + j] = 0.0;
	  for (k = 0; k < NK; ++k)
	    {
	      C[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
	    }
	}
    }

  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NL; j++)
	{
	  E[i * NL + j] = 0.0;
	  for (k = 0; k < NJ; ++k)
	    {
	      E[i * NL + j] += C[i * NJ + k] * D[k * NL + j];
	    }
	}
    }
}

void
mm2Cuda (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E,
	 DATA_TYPE* E_outputFromGpu)
{
  double t_start, t_end;

  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;
  DATA_TYPE *C_gpu;
  DATA_TYPE *D_gpu;
  DATA_TYPE *E_gpu;

  hipMalloc ((void **) &A_gpu, sizeof(DATA_TYPE) * NI * NK);
  hipMalloc ((void **) &B_gpu, sizeof(DATA_TYPE) * NK * NJ);
  hipMalloc ((void **) &C_gpu, sizeof(DATA_TYPE) * NI * NJ);
  hipMalloc ((void **) &D_gpu, sizeof(DATA_TYPE) * NJ * NL);
  hipMalloc ((void **) &E_gpu, sizeof(DATA_TYPE) * NI * NL);

  hipMemcpy (A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
  hipMemcpy (B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
  hipMemcpy (C_gpu, C, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
  hipMemcpy (D_gpu, D, sizeof(DATA_TYPE) * NJ * NL, hipMemcpyHostToDevice);
  hipMemcpy (E_gpu, E, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);

  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid1 ((size_t) ceil (((float) NJ) / ((float) block.x)),
	      (size_t) ceil (((float) NI) / ((float) block.y)));
  dim3 grid2 ((size_t) ceil (((float) NL) / ((float) block.x)),
	      (size_t) ceil (((float) NI) / ((float) block.y)));
//  t_start = rtclock ();

  cuda_timer t_mm2_k1, t_mm2_k2;
  cuda_timer_init (t_mm2_k1);
  cuda_timer_init (t_mm2_k2);
  cuda_timer_record_start (t_mm2_k1);
  mm2_kernel1 <<<grid1, block>>> (A_gpu, B_gpu, C_gpu, NI, NJ, NK, NL);
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t_mm2_k1);
  hipDeviceSynchronize ();

  cuda_timer_record_start (t_mm2_k2);
  mm2_kernel2 <<<grid2, block>>> (C_gpu, D_gpu, E_gpu, NI, NJ, NK, NL);
  cudaCheckKernel ()
  ;
  cuda_timer_record_stop (t_mm2_k2);
  hipDeviceSynchronize ();

  cuda_timer_record_get_elapsed_time (t_mm2_k1);
  cuda_timer_record_get_elapsed_time (t_mm2_k2);

//  t_end = rtclock ();
//  fprintf (stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

  hipMemcpy (E_outputFromGpu, E_gpu, sizeof(DATA_TYPE) * NI * NL,
	      hipMemcpyDeviceToHost);

  printf (
      "[trace: n=%d, bx=%d, by=%d, elapsed_mm2_kernel1=%0.4f (ms), elapsed_mm2_kernel2=%0.4f (ms)] ... ",
      NI, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t_mm2_k1.elapsed_time,
      t_mm2_k2.elapsed_time);
  hipFree (A_gpu);
  hipFree (B_gpu);
  hipFree (C_gpu);
  hipFree (D_gpu);
  hipFree (E_gpu);

  cuda_timer_destroy (t_mm2_k1);
  cuda_timer_destroy (t_mm2_k2);

}

int
main (int argc, char** argv)
{
//	double t_start, t_end;

  int n = 256, bx = 32, by = 8;

  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);

  NI = NJ = NK = NL = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;

  DATA_TYPE* C;
  DATA_TYPE* A;
  DATA_TYPE* B;
  DATA_TYPE* D;
  DATA_TYPE* E;
  DATA_TYPE* E_outputFromGpu;

  C = (DATA_TYPE*) malloc (NI * NJ * sizeof(DATA_TYPE));
  A = (DATA_TYPE*) malloc (NI * NK * sizeof(DATA_TYPE));
  B = (DATA_TYPE*) malloc (NK * NJ * sizeof(DATA_TYPE));
  D = (DATA_TYPE*) malloc (NJ * NL * sizeof(DATA_TYPE));
  E = (DATA_TYPE*) malloc (NI * NL * sizeof(DATA_TYPE));
  E_outputFromGpu = (DATA_TYPE*) malloc (NI * NL * sizeof(DATA_TYPE));

  init_array (A, B, C, D);
  GPU_argv_init ();

#pragma START_TRACING
  mm2Cuda (A, B, C, D, E, E_outputFromGpu);
#pragma STOP_TRACING

//	t_start = rtclock();
  mm2_cpu (A, B, C, D, E);
//	t_end = rtclock();
//	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

  int s = compareResults (E, E_outputFromGpu);
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");

  free (C);
  free (A);
  free (B);
  free (D);
  free (E);
  free (E_outputFromGpu);

  return 0;
}

