#include "hip/hip_runtime.h"

#include "2mm_utils.h"

///////////////////////////////////////
/////// AUTOMATICALLY ANNOTATED ///////
///////////////////////////////////////
#include "kernel_invoker.h"
///////////////////////////////////////
///////////////////////////////////////
const int kernel_info_size_param_idx_mm2_kernel1 __attribute__((used))  = 3;
const int kernel_info_size_param_idx_mm2_kernel2 __attribute__((used))  = 3;
const int kernel_info_dim_mm2_kernel1 __attribute__((used))  = 2;
const int kernel_info_dim_mm2_kernel2 __attribute__((used))  = 2;
void
init_array (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D)
{
  int i, j;
  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NK; j++)
 {
   A[i * NI + j] = ((DATA_TYPE) i * j) / NI;
 }
    }
  for (i = 0; i < NK; i++)
    {
      for (j = 0; j < NJ; j++)
 {
   B[i * NK + j] = ((DATA_TYPE) i * (j + 1)) / NJ;
 }
    }
  for (i = 0; i < NL; i++)
    {
      for (j = 0; j < NJ; j++)
 {
   C[i * NL + j] = ((DATA_TYPE) i * (j + 3)) / NL;
 }
    }
  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NL; j++)
 {
   D[i * NL + j] = ((DATA_TYPE) i * (j + 2)) / NK;
 }
    }
}
int
compareResults (DATA_TYPE *E, DATA_TYPE *E_outputFromGpu)
{
  int i, j, fail;
  fail = 0;
  for (i = 0; i < NL; i++)
    {
      for (j = 0; j < NI; j++)
 {
   if (percentDiff (
       E[i * NI + j],       E_outputFromGpu[i * NI + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
     {
       fail++;
       return (EXIT_FAILURE);
     }
 }
    }
  return EXIT_SUCCESS;
  
}
void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);
}
//__global__ void
//mm2_kernel1 (DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C, int NI, int NJ, int NK,      int NL)
//{
//  int j = blockIdx.x * blockDim.x + threadIdx.x;
//  int i = blockIdx.y * blockDim.y + threadIdx.y;
//  if ((i < NI) && (j < NJ))
//    {
//      int k;
//      for (k = 0; k < NK; k++)
// {
//   C[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
// }
//    }
//}
//__global__ void
//mm2_kernel2 (DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *E, int NI, int NJ, int NK,      int NL)
//{
//  int j = blockIdx.x * blockDim.x + threadIdx.x;
//  int i = blockIdx.y * blockDim.y + threadIdx.y;
//  if ((i < NI) && (j < NL))
//    {
//      int k;
//      for (k = 0; k < NJ; k++)
// {
//   E[i * NL + j] += C[i * NJ + k] * D[k * NL + j];
// }
//    }
//}
void
mm2_cpu (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E)
{
  int i, j, k;
  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NJ; j++)
 {
   C[i * NJ + j] = 0.0;
   for (k = 0; k < NK; ++k)
     {
       C[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
     }
 }
    }
  for (i = 0; i < NI; i++)
    {
      for (j = 0; j < NL; j++)
 {
   E[i * NL + j] = 0.0;
   for (k = 0; k < NJ; ++k)
     {
       E[i * NL + j] += C[i * NJ + k] * D[k * NL + j];
     }
 }
    }
}
void
mm2Cuda (DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E,  DATA_TYPE* E_outputFromGpu)
{
  double t_start, t_end;
  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;
  DATA_TYPE *C_gpu;
  DATA_TYPE *D_gpu;
  DATA_TYPE *E_gpu;
  hipMalloc ((void **) &A_gpu, sizeof(DATA_TYPE) * NI * NK);
  hipMalloc ((void **) &B_gpu, sizeof(DATA_TYPE) * NK * NJ);
  hipMalloc ((void **) &C_gpu, sizeof(DATA_TYPE) * NI * NJ);
  hipMalloc ((void **) &D_gpu, sizeof(DATA_TYPE) * NJ * NL);
  hipMalloc ((void **) &E_gpu, sizeof(DATA_TYPE) * NI * NL);
  hipMemcpy (A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
  hipMemcpy (B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
  hipMemcpy (C_gpu, C, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
  hipMemcpy (D_gpu, D, sizeof(DATA_TYPE) * NJ * NL, hipMemcpyHostToDevice);
  hipMemcpy (E_gpu, E, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);
  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid1 ((size_t) ceil (((float) NJ) / ((float) block.x)),       (size_t) ceil (((float) NI) / ((float) block.y)));
  dim3 grid2 ((size_t) ceil (((float) NL) / ((float) block.x)),       (size_t) ceil (((float) NI) / ((float) block.y)));
  cuda_timer t_mm2_k1, t_mm2_k2;
  cuda_timer_init (t_mm2_k1);
  cuda_timer_init (t_mm2_k2);
  cuda_timer_record_start (t_mm2_k1);
  	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_mm2_kernel1_0_name[] = "kernel_mm2_kernel1_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_mm2_kernel1_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_mm2_kernel1_sm_75_0_launch_params, grid1, block);
 
 void * kernel_mm2_kernel1_sm_75_0_kernel_params[]={&A_gpu , &B_gpu , &C_gpu , &NI , &NJ , &NK , &NL};
 
 kernel_invoker(kernel_mm2_kernel1_0_name, kernel_mm2_kernel1_sm_75_0_launch_params, kernel_mm2_kernel1_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t_mm2_k1);
  hipDeviceSynchronize ();
  cuda_timer_record_start (t_mm2_k2);
  	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_mm2_kernel2_0_name[] = "kernel_mm2_kernel2_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_mm2_kernel2_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_mm2_kernel2_sm_75_0_launch_params, grid2, block);
 
 void * kernel_mm2_kernel2_sm_75_0_kernel_params[]={&C_gpu , &D_gpu , &E_gpu , &NI , &NJ , &NK , &NL};
 
 kernel_invoker(kernel_mm2_kernel2_0_name, kernel_mm2_kernel2_sm_75_0_launch_params, kernel_mm2_kernel2_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
  cudaCheckKernel ()
  ;
  cuda_timer_record_stop (t_mm2_k2);
  hipDeviceSynchronize ();
  cuda_timer_record_get_elapsed_time (t_mm2_k1);
  cuda_timer_record_get_elapsed_time (t_mm2_k2);
  hipMemcpy (E_outputFromGpu, E_gpu, sizeof(DATA_TYPE) * NI * NL,       hipMemcpyDeviceToHost);
  printf (
      "[trace: n=%d, bx=%d, by=%d, elapsed_mm2_kernel1=%0.4f (ms), elapsed_mm2_kernel2=%0.4f (ms)] ... ",      NI, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t_mm2_k1.elapsed_time,      t_mm2_k2.elapsed_time);
  hipFree (A_gpu);
  hipFree (B_gpu);
  hipFree (C_gpu);
  hipFree (D_gpu);
  hipFree (E_gpu);
  cuda_timer_destroy (t_mm2_k1);
  cuda_timer_destroy (t_mm2_k2);
}
int
main (int argc, char** argv)
{
  int n = 256, bx = 32, by = 8;
  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);
  NI = NJ = NK = NL = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;
  DATA_TYPE* C;
  DATA_TYPE* A;
  DATA_TYPE* B;
  DATA_TYPE* D;
  DATA_TYPE* E;
  DATA_TYPE* E_outputFromGpu;
  C = (DATA_TYPE*) malloc (NI * NJ * sizeof(DATA_TYPE));
  A = (DATA_TYPE*) malloc (NI * NK * sizeof(DATA_TYPE));
  B = (DATA_TYPE*) malloc (NK * NJ * sizeof(DATA_TYPE));
  D = (DATA_TYPE*) malloc (NJ * NL * sizeof(DATA_TYPE));
  E = (DATA_TYPE*) malloc (NI * NL * sizeof(DATA_TYPE));
  E_outputFromGpu = (DATA_TYPE*) malloc (NI * NL * sizeof(DATA_TYPE));
  init_array (A, B, C, D);
  GPU_argv_init ();
#pragma START_TRACING
  mm2Cuda (A, B, C, D, E, E_outputFromGpu);
#pragma STOP_TRACING
  mm2_cpu (A, B, C, D, E);
  int s = compareResults (E, E_outputFromGpu);
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");
  free (C);
  free (A);
  free (B);
  free (D);
  free (E);
  free (E_outputFromGpu);
  return 0;
}
