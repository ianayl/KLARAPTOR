#include "hip/hip_runtime.h"
///**
// * mvt.cu: This file is part of the PolyBench/GPU 1.0 test suite.
// *
// *
// * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
// * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
// * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
// */
//

#include "mvt_utils.h"

#pragma kernel_info_size_param_idx_mvt_kernel1 = 3;
#pragma kernel_info_dim_mvt_kernel1 = 2;

#pragma kernel_info_size_param_idx_mvt_kernel2 = 3;
#pragma kernel_info_dim_mvt_kernel2 = 2;

void
init_array (DATA_TYPE* A, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y1,
	    DATA_TYPE* y2)
{
  int i, j;

  for (i = 0; i < N; i++)
    {
      x1[i] = ((DATA_TYPE) i) / N;
      x2[i] = ((DATA_TYPE) i + 1) / N;
      y1[i] = ((DATA_TYPE) i + 3) / N;
      y2[i] = ((DATA_TYPE) i + 4) / N;
      for (j = 0; j < N; j++)
	{
	  A[i * N + j] = ((DATA_TYPE) i * j) / N;
	}
    }
}

void
runMvt (DATA_TYPE* a, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y1,
	DATA_TYPE* y2)
{
  int i, j;

  for (i = 0; i < N; i++)
    {
      for (j = 0; j < N; j++)
	{
	  x1[i] = x1[i] + a[i * N + j] * y1[j];
	}
    }

  for (i = 0; i < N; i++)
    {
      for (j = 0; j < N; j++)
	{
	  x2[i] = x2[i] + a[j * N + i] * y2[j];
	}
    }
}

int
compareResults (DATA_TYPE* x1, DATA_TYPE* x1_outputFromGpu, DATA_TYPE* x2,
		DATA_TYPE* x2_outputFromGpu)
{
  int i, fail;
  fail = 0;

  for (i = 0; i < N; i++)
    {
      if (percentDiff (x1[i],
		       x1_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
	{
	  fail++;
	  return (EXIT_FAILURE);
	}

      if (percentDiff (x2[i],
		       x2_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
	{
	  fail++;
	  return (EXIT_FAILURE);
	}
    }

  return (EXIT_SUCCESS);
  // Print results
//	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
//	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);
}

__global__ void
mvt_kernel1 (DATA_TYPE *a, DATA_TYPE *x1, DATA_TYPE *y_1, int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N)
    {
      int j;
      for (j = 0; j < N; j++)
	{
	  x1[i] += a[i * N + j] * y_1[j];
	}
    }
}

__global__ void
mvt_kernel2 (DATA_TYPE *a, DATA_TYPE *x2, DATA_TYPE *y_2, int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N)
    {
      int j;
      for (j = 0; j < N; j++)
	{
	  x2[i] += a[j * N + i] * y_2[j];
	}
    }
}

void
mvtCuda (DATA_TYPE* a, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y_1,
	 DATA_TYPE* y_2, DATA_TYPE* x1_outputFromGpu,
	 DATA_TYPE* x2_outputFromGpu)
{
  double t_start, t_end;

  DATA_TYPE* a_gpu;
  DATA_TYPE* x1_gpu;
  DATA_TYPE* x2_gpu;
  DATA_TYPE* y_1_gpu;
  DATA_TYPE* y_2_gpu;

  hipMalloc ((void **) &a_gpu, sizeof(DATA_TYPE) * N * N);
  hipMalloc ((void **) &x1_gpu, sizeof(DATA_TYPE) * N);
  hipMalloc ((void **) &x2_gpu, sizeof(DATA_TYPE) * N);
  hipMalloc ((void **) &y_1_gpu, sizeof(DATA_TYPE) * N);
  hipMalloc ((void **) &y_2_gpu, sizeof(DATA_TYPE) * N);
  hipMemcpy (a_gpu, a, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);
  hipMemcpy (x1_gpu, x1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
  hipMemcpy (x2_gpu, x2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
  hipMemcpy (y_1_gpu, y_1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
  hipMemcpy (y_2_gpu, y_2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);

  dim3 block (DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid ((size_t) ceil ((float) N / ((float) DIM_THREAD_BLOCK_X)), 1);

  cuda_timer t1, t2;
  cuda_timer_init (t1);
  cuda_timer_init (t2);

//	t_start = rtclock();
  cuda_timer_record_start (t1);
  mvt_kernel1 <<<grid, block>>> (a_gpu, x1_gpu, y_1_gpu, N);
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t1);

  cuda_timer_record_start (t2);
  mvt_kernel2 <<<grid, block>>> (a_gpu, x2_gpu, y_2_gpu, N);
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t2);
  hipDeviceSynchronize ();
//	t_end = rtclock();
//	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

  hipMemcpy (x1_outputFromGpu, x1_gpu, sizeof(DATA_TYPE) * N,
	      hipMemcpyDeviceToHost);
  hipMemcpy (x2_outputFromGpu, x2_gpu, sizeof(DATA_TYPE) * N,
	      hipMemcpyDeviceToHost);

  cuda_timer_record_get_elapsed_time (t1);
  cuda_timer_record_get_elapsed_time (t2);

  printf (
      "[trace: n=%d, bx=%d, by=%d, elapsed_mvt_kernel1=%0.4f (ms), elapsed_mvt_kernel2=%0.4f (ms)] ... ",
      N, DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y, t1.elapsed_time,
      t2.elapsed_time);
  hipFree (a_gpu);
  hipFree (x1_gpu);
  hipFree (x2_gpu);
  hipFree (y_1_gpu);
  hipFree (y_2_gpu);
}

int
main (int argc, char **argv)
{
  int n = 256, bx = 32, by = 8;

  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);

  N = n;
  DIM_THREAD_BLOCK_X = bx;
  DIM_THREAD_BLOCK_Y = by;
//  double t_start, t_end;

  DATA_TYPE* a;
  DATA_TYPE* x1;
  DATA_TYPE* x2;
  DATA_TYPE* x1_outputFromGpu;
  DATA_TYPE* x2_outputFromGpu;
  DATA_TYPE* y_1;
  DATA_TYPE* y_2;

  a = (DATA_TYPE*) malloc (N * N * sizeof(DATA_TYPE));
  x1 = (DATA_TYPE*) malloc (N * sizeof(DATA_TYPE));
  x2 = (DATA_TYPE*) malloc (N * sizeof(DATA_TYPE));
  x1_outputFromGpu = (DATA_TYPE*) malloc (N * sizeof(DATA_TYPE));
  x2_outputFromGpu = (DATA_TYPE*) malloc (N * sizeof(DATA_TYPE));
  y_1 = (DATA_TYPE*) malloc (N * sizeof(DATA_TYPE));
  y_2 = (DATA_TYPE*) malloc (N * sizeof(DATA_TYPE));

  init_array (a, x1, x2, y_1, y_2);

  GPU_argv_init ();
#pragma START_TRACING
  mvtCuda (a, x1, x2, y_1, y_2, x1_outputFromGpu, x2_outputFromGpu);
#pragma STOP_TRACING
//  t_start = rtclock ();

//run the algorithm on the CPU
  runMvt (a, x1, x2, y_1, y_2);

//  t_end = rtclock ();
//  fprintf (stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

  int s = compareResults (x1, x1_outputFromGpu, x2, x2_outputFromGpu);
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");

  free (a);
  free (x1);
  free (x2);
  free (x1_outputFromGpu);
  free (x2_outputFromGpu);
  free (y_1);
  free (y_2);

  return 0;
}

