#include "hip/hip_runtime.h"

#include "covariance_utils.h"

///////////////////////////////////////
/////// AUTOMATICALLY ANNOTATED ///////
///////////////////////////////////////
#include "kernel_invoker.h"
///////////////////////////////////////
///////////////////////////////////////
const int kernel_info_size_param_idx_mean_kernel __attribute__((used))  = 2;
const int kernel_info_dim_mean_kernel __attribute__((used))  = 2;
const int kernel_info_size_param_idx_reduce_kernel __attribute__((used))  = 2;
const int kernel_info_dim_reduce_kernel __attribute__((used))  = 2;
const int kernel_info_size_param_idx_covar_kernel __attribute__((used))  = 2;
const int kernel_info_dim_covar_kernel __attribute__((used))  = 2;
void
init_arrays (DATA_TYPE* data)
{
  int i, j;
  for (i = 1; i < (M + 1); i++)
    {
      for (j = 1; j < (N + 1); j++)
 {
   data[i * (N + 1) + j] = ((DATA_TYPE) i * j) / M;
 }
    }
}
void
covariance (DATA_TYPE* data, DATA_TYPE* symmat, DATA_TYPE* mean)
{
  int i, j, j1, j2;
  
  for (j = 1; j < (M + 1); j++)
    {
      mean[j] = 0.0;
      for (i = 1; i < (N + 1); i++)
 {
   mean[j] += data[i * (M + 1) + j];
 }
      mean[j] /= FLOAT_N;
    }
  
  for (i = 1; i < (N + 1); i++)
    {
      for (j = 1; j < (M + 1); j++)
 {
   data[i * (M + 1) + j] -= mean[j];
 }
    }
  
  for (j1 = 1; j1 < (M + 1); j1++)
    {
      for (j2 = j1; j2 < (M + 1); j2++)
 {
   symmat[j1 * (M + 1) + j2] = 0.0;
   for (i = 1; i < N + 1; i++)
     {
       symmat[j1 * (M + 1) + j2] += data[i * (M + 1) + j1]
    * data[i * (M + 1) + j2];
     }
   symmat[j2 * (M + 1) + j1] = symmat[j1 * (M + 1) + j2];
 }
    }
}
int
compareResults (DATA_TYPE* symmat, DATA_TYPE* symmat_outputFromGpu)
{
  int i, j, fail;
  fail = 0;
  for (i = 1; i < (M + 1); i++)
    {
      for (j = 1; j < (N + 1); j++)
 {
   if (percentDiff (
       symmat[i * (N + 1) + j],       symmat_outputFromGpu[i * (N + 1) + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
     {
       fail++;
       return (EXIT_FAILURE);
     }
 }
    }
  return (EXIT_SUCCESS);
}
void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);
  return;
}
//__global__ void
//mean_kernel (DATA_TYPE *mean, DATA_TYPE *data, int M, int N)
//{
//  int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
//  if ((j >= 1) && (j < (M + 1)))
//    {
//      mean[j] = 0.0;
//      int i;
//      for (i = 1; i < (N + 1); i++)
// {
//   mean[j] += data[i * (M + 1) + j];
// }
//      mean[j] /= (DATA_TYPE) FLOAT_N;
//    }
//}
//__global__ void
//reduce_kernel (DATA_TYPE *mean, DATA_TYPE *data, int M, int N)
//{
//  int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
//  int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
//  if ((i >= 1) && (i < (N + 1)) && (j >= 1) && (j < (M + 1)))
//    {
//      data[i * (M + 1) + j] -= mean[j];
//    }
//}
//__global__ void
//covar_kernel (DATA_TYPE *symmat, DATA_TYPE *data, int M, int N)
//{
//  int j1 = blockIdx.x * blockDim.x + threadIdx.x + 1;
//  int i, j2;
//  if ((j1 >= 1) && (j1 < (M + 1)))
//    {
//      for (j2 = j1; j2 < (M + 1); j2++)
// {
//   symmat[j1 * (M + 1) + j2] = 0.0;
//   for (i = 1; i < (N + 1); i++)
//     {
//       symmat[j1 * (M + 1) + j2] += data[i * (M + 1) + j1]
//    * data[i * (M + 1) + j2];
//     }
//   symmat[j2 * (M + 1) + j1] = symmat[j1 * (M + 1) + j2];
// }
//    }
//}
void
covarianceCuda (DATA_TYPE* data, DATA_TYPE* symmat, DATA_TYPE* mean,  DATA_TYPE* symmat_outputFromGpu)
{
  double t_start, t_end;
  DATA_TYPE *data_gpu;
  DATA_TYPE *mean_gpu;
  DATA_TYPE *symmat_gpu;
  hipMalloc ((void **) &data_gpu, sizeof(DATA_TYPE) * (M + 1) * (N + 1));
  hipMalloc ((void **) &symmat_gpu, sizeof(DATA_TYPE) * (M + 1) * (M + 1));
  hipMalloc ((void **) &mean_gpu, sizeof(DATA_TYPE) * (M + 1));
  hipMemcpy (data_gpu, data, sizeof(DATA_TYPE) * (M + 1) * (N + 1),       hipMemcpyHostToDevice);
  hipMemcpy (symmat_gpu, symmat, sizeof(DATA_TYPE) * (M + 1) * (M + 1),       hipMemcpyHostToDevice);
  hipMemcpy (mean_gpu, mean, sizeof(DATA_TYPE) * (M + 1),       hipMemcpyHostToDevice);
  dim3 block1 (DIM_THREAD_BLOCK_KERNEL_1_X, DIM_THREAD_BLOCK_KERNEL_1_Y);
  dim3 grid1 (
      (size_t) (ceil ((float) M) / ((float) DIM_THREAD_BLOCK_KERNEL_1_X)), 1);
  dim3 block2 (DIM_THREAD_BLOCK_KERNEL_2_X, DIM_THREAD_BLOCK_KERNEL_2_Y);
  dim3 grid2 (
      (size_t) (ceil ((float) M) / ((float) DIM_THREAD_BLOCK_KERNEL_2_X)),      (size_t) (ceil ((float) N) / ((float) DIM_THREAD_BLOCK_KERNEL_2_X)));
  dim3 block3 (DIM_THREAD_BLOCK_KERNEL_3_X, DIM_THREAD_BLOCK_KERNEL_3_Y);
  dim3 grid3 (
      (size_t) (ceil ((float) M) / ((float) DIM_THREAD_BLOCK_KERNEL_3_X)), 1);
  cuda_timer t1, t2, t3;
  cuda_timer_init (t1);
  cuda_timer_init (t2);
  cuda_timer_init (t3);
  cuda_timer_record_start (t1);
  	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_mean_kernel_0_name[] = "kernel_mean_kernel_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_mean_kernel_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_mean_kernel_sm_75_0_launch_params, grid1, block1);
 
 void * kernel_mean_kernel_sm_75_0_kernel_params[]={&mean_gpu , &data_gpu , &M , &N};
 
 kernel_invoker(kernel_mean_kernel_0_name, kernel_mean_kernel_sm_75_0_launch_params, kernel_mean_kernel_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
  cudaCheckKernel ()
  ;
  cuda_timer_record_stop (t1);
  hipDeviceSynchronize ();
  cuda_timer_record_start (t2);
  	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_reduce_kernel_0_name[] = "kernel_reduce_kernel_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_reduce_kernel_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_reduce_kernel_sm_75_0_launch_params, grid2, block2);
 
 void * kernel_reduce_kernel_sm_75_0_kernel_params[]={&mean_gpu , &data_gpu , &M , &N};
 
 kernel_invoker(kernel_reduce_kernel_0_name, kernel_reduce_kernel_sm_75_0_launch_params, kernel_reduce_kernel_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
  cudaCheckKernel ()
  ;
  cuda_timer_record_stop (t2);
  hipDeviceSynchronize ();
  cuda_timer_record_start (t3);
  	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_covar_kernel_0_name[] = "kernel_covar_kernel_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_covar_kernel_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_covar_kernel_sm_75_0_launch_params, grid3, block3);
 
 void * kernel_covar_kernel_sm_75_0_kernel_params[]={&symmat_gpu , &data_gpu , &M , &N};
 
 kernel_invoker(kernel_covar_kernel_0_name, kernel_covar_kernel_sm_75_0_launch_params, kernel_covar_kernel_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
  cudaCheckKernel ()
  ;
  cuda_timer_record_stop (t3);
  hipDeviceSynchronize ();
  hipMemcpy (symmat_outputFromGpu, symmat_gpu,       sizeof(DATA_TYPE) * (M + 1) * (N + 1), hipMemcpyDeviceToHost);
  cuda_timer_record_get_elapsed_time(t1);
  cuda_timer_record_get_elapsed_time(t2);
  cuda_timer_record_get_elapsed_time(t3);
  printf ("[trace: n=%d, bx=%d, by=%d, "
   "elapsed_mean_kernel=%0.4f (ms),\n"
   " elapsed_reduce_kernel=%0.4f (ms), "
   " elapsed_covar_kernel=%0.4f (ms)] ... ",   M, DIM_THREAD_BLOCK_KERNEL_3_X, DIM_THREAD_BLOCK_KERNEL_3_Y,   t1.elapsed_time, t2.elapsed_time, t3.elapsed_time);
  hipFree (data_gpu);
  hipFree (symmat_gpu);
  hipFree (mean_gpu);
  cuda_timer_destroy (t1);
  cuda_timer_destroy (t2);
  cuda_timer_destroy (t3);
}
int
main (int argc, char** argv)
{
  DATA_TYPE* data;
  DATA_TYPE* symmat;
  DATA_TYPE* mean;
  DATA_TYPE* symmat_outputFromGpu;
  int n = 256, bx = 32, by = 8;
  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);
  M = N = n;
  DIM_THREAD_BLOCK_KERNEL_1_X = bx;
  DIM_THREAD_BLOCK_KERNEL_1_Y = by;
  DIM_THREAD_BLOCK_KERNEL_2_X = bx;
  DIM_THREAD_BLOCK_KERNEL_2_Y = by;
  DIM_THREAD_BLOCK_KERNEL_3_X = bx;
  DIM_THREAD_BLOCK_KERNEL_3_Y = by;
  data = (DATA_TYPE*) malloc ((M + 1) * (N + 1) * sizeof(DATA_TYPE));
  symmat = (DATA_TYPE*) malloc ((M + 1) * (M + 1) * sizeof(DATA_TYPE));
  mean = (DATA_TYPE*) malloc ((M + 1) * sizeof(DATA_TYPE));
  symmat_outputFromGpu = (DATA_TYPE*) malloc (
      (M + 1) * (M + 1) * sizeof(DATA_TYPE));
  init_arrays (data);
  GPU_argv_init ();
#pragma START_TRACING
  covarianceCuda (data, symmat, mean, symmat_outputFromGpu);
#pragma STOP_TRACING
  covariance (data, symmat, mean);
  int s = compareResults (symmat, symmat_outputFromGpu);
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");
  free (data);
  free (symmat);
  free (mean);
  free (symmat_outputFromGpu);
  return 0;
}
