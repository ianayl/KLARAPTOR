#include "hip/hip_runtime.h"

#include "correlation_utils.h"

///////////////////////////////////////
/////// AUTOMATICALLY ANNOTATED ///////
///////////////////////////////////////
#include "kernel_invoker.h"
///////////////////////////////////////
///////////////////////////////////////
const int kernel_info_size_param_idx_mean_kernel __attribute__((used))  = 2;
const int kernel_info_dim_mean_kernel __attribute__((used))  = 2;
const int kernel_info_size_param_idx_std_kernel __attribute__((used))  = 3;
const int kernel_info_dim_std_kernel __attribute__((used))  = 2;
const int kernel_info_size_param_idx_reduce_kernel __attribute__((used))  = 3;
const int kernel_info_dim_reduce_kernel __attribute__((used))  = 2;
const int kernel_info_size_param_idx_corr_kernel __attribute__((used))  = 2;
const int kernel_info_dim_corr_kernel __attribute__((used))  = 2;
void
init_arrays (DATA_TYPE* data)
{
  int i, j;
  for (i = 0; i < (M + 1); i++)
    {
      for (j = 0; j < (N + 1); j++)
 {
   data[i * (N + 1) + j] = ((DATA_TYPE) i * j) / (M + 1);
 }
    }
}
void
correlation (DATA_TYPE* data, DATA_TYPE* mean, DATA_TYPE* stddev,      DATA_TYPE* symmat)
{
  int i, j, j1, j2;
  
  for (j = 1; j < (M + 1); j++)
    {
      mean[j] = 0.0;
      for (i = 1; i < (N + 1); i++)
 {
   mean[j] += data[i * (M + 1) + j];
 }
      mean[j] /= (DATA_TYPE) FLOAT_N;
    }
  
  for (j = 1; j < (M + 1); j++)
    {
      stddev[j] = 0.0;
      for (i = 1; i < (N + 1); i++)
 {
   stddev[j] += (data[i * (M + 1) + j] - mean[j])
       * (data[i * (M + 1) + j] - mean[j]);
 }
      stddev[j] /= FLOAT_N;
      stddev[j] = sqrt_of_array_cell(stddev, j);
      stddev[j] = stddev[j] <= EPS ? 1.0 : stddev[j];
    }
  
  for (i = 1; i < (N + 1); i++)
    {
      for (j = 1; j < (M + 1); j++)
 {
   data[i * (M + 1) + j] -= mean[j];
   data[i * (M + 1) + j] /= (sqrt (FLOAT_N) * stddev[j]);
 }
    }
  
  for (j1 = 1; j1 < M; j1++)
    {
      symmat[j1 * (M + 1) + j1] = 1.0;
      for (j2 = j1 + 1; j2 < (M + 1); j2++)
 {
   symmat[j1 * (M + 1) + j2] = 0.0;
   for (i = 1; i < (N + 1); i++)
     {
       symmat[j1 * (M + 1) + j2] += (data[i * (M + 1) + j1]
    * data[i * (M + 1) + j2]);
     }
   symmat[j2 * (M + 1) + j1] = symmat[j1 * (M + 1) + j2];
 }
    }
  symmat[M * (M + 1) + M] = 1.0;
}
int
compareResults (DATA_TYPE* symmat, DATA_TYPE* symmat_outputFromGpu)
{
  int i, j, fail;
  fail = 0;
  for (i = 1; i < (M + 1); i++)
    {
      for (j = 1; j < (N + 1); j++)
 {
   if (percentDiff (
       symmat[i * (N + 1) + j],       symmat_outputFromGpu[i * (N + 1) + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
     {
       fail++;
       printf ("i: %d j: %d\n1: %f 2: %f\n", i, j, symmat[i * N + j],        symmat_outputFromGpu[i * N + j]);
       return (EXIT_FAILURE);
     }
 }
    }
  
  return (EXIT_SUCCESS);
}
void
GPU_argv_init ()
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties (&deviceProp, GPU_DEVICE);
  printf ("[running on device %d: %s]\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice ( GPU_DEVICE);
}
//__global__ void
//mean_kernel (DATA_TYPE *mean, DATA_TYPE *data, int M, int N)
//{
//  int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
//  if ((j >= 1) && (j < (M + 1)))
//    {
//      mean[j] = 0.0;
//      int i;
//      for (i = 1; i < (N + 1); i++)
// {
//   mean[j] += data[i * (M + 1) + j];
// }
//      mean[j] /= (DATA_TYPE) FLOAT_N;
//    }
//}
//__global__ void
//std_kernel (DATA_TYPE *mean, DATA_TYPE *std, DATA_TYPE *data, int M, int N)
//{
//  int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
//  if ((j >= 1) && (j < (M + 1)))
//    {
//      std[j] = 0.0;
//      int i;
//      for (i = 1; i < (N + 1); i++)
// {
//   std[j] += (data[i * (M + 1) + j] - mean[j])
//       * (data[i * (M + 1) + j] - mean[j]);
// }
//      std[j] /= (FLOAT_N);
//      std[j] = sqrt (std[j]);
//      if (std[j] <= EPS)
// {
//   std[j] = 1.0;
// }
//    }
//}
//__global__ void
//reduce_kernel (DATA_TYPE *mean, DATA_TYPE *std, DATA_TYPE *data, int M, int N)
//{
//  int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
//  int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
//  if ((i >= 1) && (i < (N + 1)) && (j >= 1) && (j < (M + 1)))
//    {
//      data[i * (M + 1) + j] -= mean[j];
//      data[i * (M + 1) + j] /= (sqrt (FLOAT_N) * std[j]);
//    }
//}
//__global__ void
//corr_kernel (DATA_TYPE *symmat, DATA_TYPE *data, int M, int N)
//{
//  int j1 = blockIdx.x * blockDim.x + threadIdx.x + 1;
//  int i, j2;
//  if ((j1 >= 1) && (j1 < M))
//    {
//      symmat[j1 * (M + 1) + j1] = 1.0;
//      for (j2 = (j1 + 1); j2 < (M + 1); j2++)
// {
//   symmat[j1 * (M + 1) + j2] = 0.0;
//   for (i = 1; i < (N + 1); i++)
//     {
//       symmat[j1 * (M + 1) + j2] += data[i * (M + 1) + j1]
//    * data[i * (M + 1) + j2];
//     }
//   symmat[j2 * (M + 1) + j1] = symmat[j1 * (M + 1) + j2];
// }
//    }
//}
void
correlationCuda (DATA_TYPE* data, DATA_TYPE* mean, DATA_TYPE* stddev,   DATA_TYPE* symmat, DATA_TYPE* symmat_outputFromGpu)
{
  double t_start, t_end;
  DATA_TYPE *data_gpu;
  DATA_TYPE *stddev_gpu;
  DATA_TYPE *mean_gpu;
  DATA_TYPE *symmat_gpu;
  hipMalloc ((void **) &data_gpu, sizeof(DATA_TYPE) * (M + 1) * (N + 1));
  hipMalloc ((void **) &symmat_gpu, sizeof(DATA_TYPE) * (M + 1) * (N + 1));
  hipMalloc ((void **) &stddev_gpu, sizeof(DATA_TYPE) * (M + 1));
  hipMalloc ((void **) &mean_gpu, sizeof(DATA_TYPE) * (M + 1));
  hipMemcpy (data_gpu, data, sizeof(DATA_TYPE) * (M + 1) * (N + 1),       hipMemcpyHostToDevice);
  hipMemcpy (symmat_gpu, symmat, sizeof(DATA_TYPE) * (M + 1) * (N + 1),       hipMemcpyHostToDevice);
  hipMemcpy (stddev_gpu, stddev, sizeof(DATA_TYPE) * (M + 1),       hipMemcpyHostToDevice);
  hipMemcpy (mean_gpu, mean, sizeof(DATA_TYPE) * (M + 1),       hipMemcpyHostToDevice);
  dim3 block1 (DIM_THREAD_BLOCK_KERNEL_1_X, DIM_THREAD_BLOCK_KERNEL_1_Y);
  dim3 grid1 (
      (size_t) (ceil ((float) (M)) / ((float) DIM_THREAD_BLOCK_KERNEL_1_X)), 1);
  dim3 block2 (DIM_THREAD_BLOCK_KERNEL_2_X, DIM_THREAD_BLOCK_KERNEL_2_Y);
  dim3 grid2 (
      (size_t) (ceil ((float) (M)) / ((float) DIM_THREAD_BLOCK_KERNEL_2_X)), 1);
  dim3 block3 (DIM_THREAD_BLOCK_KERNEL_3_X, DIM_THREAD_BLOCK_KERNEL_3_Y);
  dim3 grid3 (
      (size_t) (ceil ((float) (M)) / ((float) DIM_THREAD_BLOCK_KERNEL_3_X)),      (size_t) (ceil ((float) (N)) / ((float) DIM_THREAD_BLOCK_KERNEL_3_Y)));
  dim3 block4 (DIM_THREAD_BLOCK_KERNEL_4_X, DIM_THREAD_BLOCK_KERNEL_4_Y);
  dim3 grid4 (
      (size_t) (ceil ((float) (M)) / ((float) DIM_THREAD_BLOCK_KERNEL_4_X)), 1);
  cuda_timer t1, t2, t3, t4;
  cuda_timer_init (t1);
  cuda_timer_init (t2);
  cuda_timer_init (t3);
  cuda_timer_init (t4);
  cuda_timer_record_start (t1);
  	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_mean_kernel_0_name[] = "kernel_mean_kernel_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_mean_kernel_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_mean_kernel_sm_75_0_launch_params, grid1, block1);
 
 void * kernel_mean_kernel_sm_75_0_kernel_params[]={&mean_gpu , &data_gpu , &M , &N};
 
 kernel_invoker(kernel_mean_kernel_0_name, kernel_mean_kernel_sm_75_0_launch_params, kernel_mean_kernel_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t1);
  hipDeviceSynchronize ();
  cuda_timer_record_start (t2);
  	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_std_kernel_0_name[] = "kernel_std_kernel_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_std_kernel_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_std_kernel_sm_75_0_launch_params, grid2, block2);
 
 void * kernel_std_kernel_sm_75_0_kernel_params[]={&mean_gpu , &stddev_gpu , &data_gpu , &M , &N};
 
 kernel_invoker(kernel_std_kernel_0_name, kernel_std_kernel_sm_75_0_launch_params, kernel_std_kernel_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t2);
  hipDeviceSynchronize ();
  cuda_timer_record_start (t3);
  	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_reduce_kernel_0_name[] = "kernel_reduce_kernel_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_reduce_kernel_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_reduce_kernel_sm_75_0_launch_params, grid3, block3);
 
 void * kernel_reduce_kernel_sm_75_0_kernel_params[]={&mean_gpu , &stddev_gpu , &data_gpu , &M , &N};
 
 kernel_invoker(kernel_reduce_kernel_0_name, kernel_reduce_kernel_sm_75_0_launch_params, kernel_reduce_kernel_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t3);
  hipDeviceSynchronize ();
  cuda_timer_record_start (t4);
  	
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION BEGINS HERE /////////
 ////////////////////////////////////////////////////////////////////////
	
 

 char kernel_corr_kernel_0_name[] = "kernel_corr_kernel_sm_75";
 
 //launch_params: 3 for grid_dim, 3 for block_dim, 1 for dynamic_shared_mem_bytes;
 int kernel_corr_kernel_sm_75_0_launch_params[6];
 set_kernel_launch_params(kernel_corr_kernel_sm_75_0_launch_params, grid4, block4);
 
 void * kernel_corr_kernel_sm_75_0_kernel_params[]={&symmat_gpu , &data_gpu , &M , &N};
 
 kernel_invoker(kernel_corr_kernel_0_name, kernel_corr_kernel_sm_75_0_launch_params, kernel_corr_kernel_sm_75_0_kernel_params);
 
 ////////////////////////////////////////////////////////////////////////
 ////////// WARNING: AUTOMATICALLY ANNOTATED REGION ENDS HERE ///////////
 ////////////////////////////////////////////////////////////////////////
	
  cudaCheckKernel()
  ;
  cuda_timer_record_stop (t4);
  hipDeviceSynchronize ();
  DATA_TYPE valueAtSymmatIndexMTimesMPlus1PlusMPoint = 1.0;
  hipMemcpy (&(symmat_gpu[(M) * (M + 1) + (M)]),       &valueAtSymmatIndexMTimesMPlus1PlusMPoint, sizeof(DATA_TYPE),       hipMemcpyHostToDevice);
  hipMemcpy (symmat_outputFromGpu, symmat_gpu,       sizeof(DATA_TYPE) * (M + 1) * (N + 1), hipMemcpyDeviceToHost);
  cuda_timer_record_get_elapsed_time (t1);
  cuda_timer_record_get_elapsed_time (t2);
  cuda_timer_record_get_elapsed_time (t3);
  cuda_timer_record_get_elapsed_time (t4);
  printf ("[trace: n=%d, bx=%d, by=%d, "
   "elapsed_mean_kernel=%0.4f (ms),\n"
   " elapsed_std_kernel=%0.4f (ms), "
   " elapsed_reduce_kernel=%0.4f (ms),\n"
   " elapsed_corr_kernel=%0.4f (ms)] ... ",   M, DIM_THREAD_BLOCK_KERNEL_4_X, DIM_THREAD_BLOCK_KERNEL_4_Y,   t1.elapsed_time, t2.elapsed_time, t3.elapsed_time, t4.elapsed_time);
  hipFree (data_gpu);
  hipFree (symmat_gpu);
  hipFree (stddev_gpu);
  hipFree (mean_gpu);
}
int
main (int argc, char ** argv)
{
  int n = 256, bx = 32, by = 8;
  if (argc > 1)
    n = atoi (argv[1]);
  if (argc > 2)
    bx = atoi (argv[2]);
  if (argc > 3)
    by = atoi (argv[3]);
  M = N = n;
  int bxby = bx * by;
  DIM_THREAD_BLOCK_KERNEL_1_X = bx;
  DIM_THREAD_BLOCK_KERNEL_1_Y = by;
  DIM_THREAD_BLOCK_KERNEL_2_X = bx;
  DIM_THREAD_BLOCK_KERNEL_2_Y = by;
  DIM_THREAD_BLOCK_KERNEL_4_X = bx;
  DIM_THREAD_BLOCK_KERNEL_4_Y = by;
  DIM_THREAD_BLOCK_KERNEL_3_X = bx;
  DIM_THREAD_BLOCK_KERNEL_3_X = by;
  DATA_TYPE* data;
  DATA_TYPE* mean;
  DATA_TYPE* stddev;
  DATA_TYPE* symmat;
  DATA_TYPE* symmat_outputFromGpu;
  data = (DATA_TYPE*) malloc ((M + 1) * (N + 1) * sizeof(DATA_TYPE));
  mean = (DATA_TYPE*) malloc ((M + 1) * sizeof(DATA_TYPE));
  stddev = (DATA_TYPE*) malloc ((M + 1) * sizeof(DATA_TYPE));
  symmat = (DATA_TYPE*) malloc ((M + 1) * (N + 1) * sizeof(DATA_TYPE));
  symmat_outputFromGpu = (DATA_TYPE*) malloc (
      (M + 1) * (N + 1) * sizeof(DATA_TYPE));
  init_arrays (data);
  GPU_argv_init ();
#pragma START_TRACING
  correlationCuda (data, mean, stddev, symmat, symmat_outputFromGpu);
#pragma STOP_TRACING
  correlation (data, mean, stddev, symmat);
  int s = compareResults (symmat, symmat_outputFromGpu);
  if (s == EXIT_SUCCESS)
    printf ("PASS\n");
  else
    printf ("FAIL\n");
  free (data);
  free (mean);
  free (stddev);
  free (symmat);
  free (symmat_outputFromGpu);
  return 0;
}
