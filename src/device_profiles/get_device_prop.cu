#include "hip/hip_runtime.h"
/* This sample queries the properties of the CUDA devices
 * present in the system via CUDA Runtime API.
 * modified from CUDA 7 sdk*/

#include <stdio.h>
#include <ctype.h>
#include <hip/hip_runtime.h> 

typedef unsigned int u32;
typedef unsigned long long int u64;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void
gpuAssert (hipError_t code, const char *file, int line, bool abort = true)
{
  if (code != hipSuccess)
    {
      fprintf (stderr, "GPUassert: %s %s %d\n", hipGetErrorString (code), file,
	       line);
      if (abort)
	exit (code);

    }
}

#if CUDART_VERSION < 5000

// CUDA-C includes
#include <hip/hip_runtime.h>

// This function wraps the CUDA Driver API into a template function
template<class T>
  inline void
  getCudaAttribute (T *attribute, hipDeviceAttribute_t device_attribute,
		    int device)
  {
    hipError_t error = hipDeviceGetAttribute (attribute, device_attribute, device);

    if (hipSuccess != error)
      {
	fprintf (
	    stderr,
	    "cuSafeCallNoSync() Driver API error = %04d from file <%s>, line %i.\n",
	    error, __FILE__, __LINE__);

	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	hipDeviceReset ();
	exit (EXIT_FAILURE);
      }
  }

#endif /* CUDART_VERSION < 5000 */

///////////////////////////////////////

/* The folloing function is copied and modified from
 * CUDA-7.0 SDK samples/common/inc/hip/hip_runtime_api.h
 * //#include <hip/hip_runtime_api.h>
 */

// Beginning of GPU Architecture definitions
inline int
_ConvertSMVer2Cores (int major, int minor)
{
  // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
  typedef struct
  {
    int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = {
        {0x30, 192},
        {0x32, 192},
        {0x35, 192},
        {0x37, 192},
        {0x50, 128},
        {0x52, 128},
        {0x53, 128},
        {0x60,  64},
        {0x61, 128},
        {0x62, 128},
        {0x70,  64},
        {0x72,  64},
        {0x75,  64},
        {0x80,  64},
        {0x86, 128},
        {0x87, 128},
        {-1, -1}};

  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1)
    {
      if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
	{
	  return nGpuArchCoresPerSM[index].Cores;
	}

      index++;
    }

  // If we don't find the values, we default use the previous one to run properly
//    printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index-1].Cores);
  return nGpuArchCoresPerSM[index - 1].Cores;
}

///////////////////////////////////////

inline int
get_nblocks_per_SM (int major, int minor)
{
  // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
  typedef struct
  {
    int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] =
    {
	  { 0x30, 16 }, // Kepler Generation (SM 3.0) GK10x class
	  { 0x32, 16 }, // Kepler Generation (SM 3.2) GK10x class
	  { 0x35, 16 }, // Kepler Generation (SM 3.5) GK11x class
	  { 0x37, 16 }, // Kepler Generation (SM 3.7) GK21x class
	  { 0x50, 32 }, // Maxwell Generation (SM 5.0) GM10x class
	  { 0x52, 32 }, // Maxwell Generation (SM 5.2) GM20x class
      { 0x53, 32 },
	  { 0x60, 32 }, // Pascal Generation (SM 6.0)  GP10X class
	  { 0x61, 32 }, // Pascal Generation (SM 6.1) GP10X class
	  { 0x62, 32 }, // Pascal Generation (SM 6.2) GP10X class
	  { 0x70, 32 }, // Volta Generation (SM 7.0) GV10x class
      { 0x72, 32 },
      { 0x75, 32 },
      { 0x80, 32 },
      { 0x86, 16 },
      { 0x87, 16 },
	  { -1, -1 } };
  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1)
    {
      if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
	{
	  return nGpuArchCoresPerSM[index].Cores;
	}

      index++;
    }

  // If we don't find the values, we default use the previous one to run properly
//    printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index-1].Cores);
  return nGpuArchCoresPerSM[index - 1].Cores;
}

///////////////////////////////////////

unsigned long long int
compute_peak_theoretical_bw_kflops (int memory_bus_width,
				    unsigned long long int memory_clock_khz)
{

//	unsigned long long int peak_bw_kflops = 2 * (memory_bus_width / 8) * memory_clock_khz;
  unsigned long long int peak_bw_kflops = (memory_bus_width >> 2)
      * memory_clock_khz;
  return peak_bw_kflops;
}

///////////////////////////////////////

char *
to_lower_str (const char * input_src_str)
{

  int n = strlen (input_src_str);

  char * src_str = (char*) malloc (n * sizeof(char));
  char * dest_str = (char*) malloc (n * sizeof(char));
  memcpy (src_str, input_src_str, n);
  int len = 0;
  int n_tokens = 0;

  for (int i = 0; i < n; i++)
    {
      if ('a' <= src_str[i] && src_str[i] <= 'z')
	continue;
      if ('A' <= src_str[i] && src_str[i] <= 'Z')
	continue;
      if ('0' <= src_str[i] && src_str[i] <= '9')
	continue;
      src_str[i] = '_';
    }

  for (int i = 0; i < n; i++)
    {
      if (src_str[i] == '_')
	{
	  if (n_tokens != 1)
	    n_tokens++;
	  continue;
	}

      if (n_tokens == 1)
	len += sprintf (dest_str + len, "%c", tolower (src_str[i]));
    }

  printf ("\ndest_str=%s\n", dest_str);
  return dest_str;

}

///////////////////////////////////////

int
host_get_device_specs_full (int argc, char ** argv)
{
//	printf("CUDA Device Query (Runtime API) version (CUDART static linking)\n");

  int n_devices = 0;
  hipError_t error_id = hipGetDeviceCount (&n_devices);

  if (error_id != hipSuccess)
    {
      printf ("hipGetDeviceCount returned %d\n-> %s\n", (int) error_id,
	      hipGetErrorString (error_id));
//		printf("Result = FAIL\n");
      exit (EXIT_FAILURE);
    }

// This function call returns 0 if there are no CUDA capable devices.
  if (n_devices == 0)
    {
      printf ("-There are no available device(s) that support CUDA\n");
      exit (EXIT_FAILURE);
    }
  else
    {
      printf ("[Found %d CUDA Capable device(s) ... ]\n", n_devices);
    }

  int dev_id, driver_version = 0, runtime_version = 0;

  for (dev_id = 0; dev_id < n_devices; ++dev_id)
    {
      char file_name[256];
      char device_name[256];

      hipSetDevice (dev_id);
      hipDeviceProp_t deviceProp;

      hipGetDeviceProperties (&deviceProp, dev_id);

      sprintf (device_name, "%s", deviceProp.name);
      printf ("checking device : %s\n", device_name);
      sprintf (file_name, "%s.full.specs", to_lower_str (device_name));
      printf ("file_name=%s\n", file_name);
      FILE* file = fopen (file_name, "w");

      fprintf (file, "[device_id: %d]\n", dev_id);
      fprintf (file, "[device_name: %s]\n", deviceProp.name);

      hipDriverGetVersion (&driver_version);
      hipRuntimeGetVersion (&runtime_version);
      fprintf (file, "[driver_version: %d.%d]\n", driver_version / 1000,
	       (driver_version % 100) / 10);
      fprintf (file, "[runtim_version: %d.%d]\n", runtime_version / 1000,
	       (runtime_version % 100) / 10);
      fprintf (file, "[compute_capability: %d.%d]\n", deviceProp.major,
	       deviceProp.minor);

      fprintf (file, "[global_memory_bytes: %llu]\n",
	       (unsigned long long) deviceProp.totalGlobalMem);
      fprintf (file, "[n_sm: %2d]\n", deviceProp.multiProcessorCount);

      fprintf (file, "[n_cores_per_sm: %3d]\n",
	       _ConvertSMVer2Cores (deviceProp.major, deviceProp.minor));

      fprintf (file, "[n_blocks_per_sm: %d]\n",
	       get_nblocks_per_SM (deviceProp.major, deviceProp.minor));

      fprintf (
	  file,
	  "[n_cores: %d]\n",
	  _ConvertSMVer2Cores (deviceProp.major, deviceProp.minor)
	      * deviceProp.multiProcessorCount);
//		fprintf(file,
//				"(%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
//				deviceProp.multiProcessorCount,
//				_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
//				_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor)
//						* deviceProp.multiProcessorCount);
      fprintf (file, "[max_clock_rate_khz: %d]\n", deviceProp.clockRate);
      fprintf (file, "[max_clock_rate_mhz: %.0f]\n",
	       deviceProp.clockRate * 1e-3f);
      fprintf (file, "[max_clock_rate_ghz: %.2f]\n",
	       deviceProp.clockRate * 1e-6f);

      int memory_clock = 0;
      int memory_bus_width = 0;
      int l2_cache_size = 0;

#if CUDART_VERSION >= 5000
// This is supported in CUDA 5.0 (runtime API device properties)
//		printf("  Memory Clock rate:                             %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
//		printf("  Memory Bus Width:                              %d-bit\n", deviceProp.memoryBusWidth);

      memory_clock=deviceProp.memoryClockRate;
      memory_bus_width=deviceProp.memoryBusWidth;
      if (deviceProp.l2CacheSize)
	{
//			printf("  L2 Cache Size:                                 %d bytes\n", deviceProp.l2CacheSize);
	  l2_cache_size=deviceProp.l2CacheSize;
	}

#else
// This only available in CUDA 4.0-4.2 (but these were only exposed in the CUDA Driver API)
//		int memoryClock;
      getCudaAttribute<int> (&memory_clock,
			     hipDeviceAttributeMemoryClockRate, dev_id);

//		int memBusWidth;
      getCudaAttribute<int> (&memory_bus_width,
			     hipDeviceAttributeMemoryBusWidth,
			     dev_id);

//		int L2CacheSize;
      getCudaAttribute<int> (&l2_cache_size, hipDeviceAttributeL2CacheSize,
			     dev_id);

#endif

      fprintf (file, "[memory_clock_rate_khz: %d]\n", memory_clock);
      fprintf (file, "[memory_clock_rate_mhz: %.0f]\n", memory_clock * 1e-3f);
      fprintf (file, "[memory_clock_rate_ghz: %.2f]\n", memory_clock * 1e-6f);

      fprintf (file, "[memory_bus_width_bits: %d]\n", memory_bus_width);
      if (l2_cache_size != 0)
	{
	  fprintf (file, "[l2_cache_size_bytes: %d]\n", l2_cache_size);
	}

      fprintf (
	  file,
	  "[peak_bw_gflops: %.2f]\n",
	  compute_peak_theoretical_bw_kflops (memory_bus_width, memory_clock)
	      * 1e-6f);

//		printf(
//				"  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n",
//				deviceProp.maxTexture1D, deviceProp.maxTexture2D[0],
//				deviceProp.maxTexture2D[1], deviceProp.maxTexture3D[0],
//				deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
//		printf(
//				"  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
//				deviceProp.maxTexture1DLayered[0],
//				deviceProp.maxTexture1DLayered[1]);
//		printf(
//				"  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d layers\n",
//				deviceProp.maxTexture2DLayered[0],
//				deviceProp.maxTexture2DLayered[1],
//				deviceProp.maxTexture2DLayered[2]);

      fprintf (file, "[total_constant memory_bytes: %lu]\n",
	       deviceProp.totalConstMem);
      fprintf (file, "[total_shared_memory_per_block_bytes: %lu]\n",
	       deviceProp.sharedMemPerBlock);
      fprintf (file, "[total_registers_available_per_block: %d]\n",
	       deviceProp.regsPerBlock);
      fprintf (file, "[warp_size: %d]\n", deviceProp.warpSize);
      fprintf (file, "[maximum_number_of_threads_per_sm: %d]\n",
	       deviceProp.maxThreadsPerMultiProcessor);
      fprintf (file, "[maximum_number_of_threads_per_block: %d]\n",
	       deviceProp.maxThreadsPerBlock);
      fprintf (file, "[max_dimensions_of_block (x,y,z): (%d, %d, %d)]\n",
	       deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
	       deviceProp.maxThreadsDim[2]);
      fprintf (file, "[max_dimensions_of_grid (x,y,z): (%d, %d, %d)]\n",
	       deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
	       deviceProp.maxGridSize[2]);
//		fprintf(file,"  Maximum memory pitch:                          %lu bytes\n",
//				deviceProp.memPitch);
//		printf("  Texture alignment:                             %lu bytes\n",
//				deviceProp.textureAlignment);
      fprintf (file, "[concurrent_copy_and_kernel_execution: %d\n",
	       deviceProp.deviceOverlap);
      fprintf (file, "[n_copy_engines: %d]\n", deviceProp.asyncEngineCount);
//		fprintf(file,"  Run time limit on kernels:                     %s\n",
//				deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
      fprintf (file, "[integrated_gpu_sharing_host_memory: %d\n",
	       deviceProp.integrated);
      fprintf (file, "[support_host_page_locked_memory_mapping: %d]\n",
	       deviceProp.canMapHostMemory);
//		fprintf(file,"[alignment_requirement_for_surfaces: %d]\n",
//				deviceProp.surfaceAlignment);
      fprintf (file, "[ecc_support: %d]\n", deviceProp.ECCEnabled);
      fprintf (file, "[support_unified_addressing (UVA): %d]\n",
	       deviceProp.unifiedAddressing);
//		printf(
//				"  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n",
//				deviceProp.pciDomainID, deviceProp.pciBusID,
//				deviceProp.pciDeviceID);

//		const char *sComputeMode[] =
//				{
//						"Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
//						"Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
//						"Prohibited (no host thread can use ::hipSetDevice() with this device)",
//						"Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
//						"Unknown",
//						NULL };
//		printf("  Compute Mode:\n");
//		printf("     < %s >\n", sComputeMode[deviceProp.computeMode]);
      fclose (file);

    }

// finish
// hipDeviceReset causes the driver to clean up all state. While
// not mandatory in normal operation, it is good practice.  It is also
// needed to ensure correct operation when the application is being
// profiled. Calling hipDeviceReset causes all profile data to be
// flushed before the application exits
  hipDeviceReset ();
  exit (EXIT_SUCCESS);
}

///////////////////////////////////////

__global__ void
kernel_gmem_latency_v0 (u32* a, u32*b, u32*clock_count, int n)
{

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	printf("tid=%d\n", tid);
//	a[tid] = 12;//b[tid]; //tid;//stop - start;
  int t0, t1;
  clock_t start, stop;
  start = clock ();
  t0 = a[tid];
  stop = clock ();
  t1 = b[tid];
  a[tid] = t0 - t1;
  clock_count[tid] = (u32 (stop - start));
//	if (tid==0)
//		printf("%llu , %llu \n", start, stop-start);

}

///////////////////////////////////////
__global__ void
kernel_gmem_non_cached_latency (u32* a, u32*b, u32*clock_count, int n)
{

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int t;
  clock_t start, elapsed = 0;

  int rep = 1;
  for (int i = 0; i < rep; i++)
    {
      start = clock ();
      t = b[tid];
      elapsed += clock () - start;
      a[tid] += t * (i + rep);
      b[tid] = a[tid];
    }

  clock_count[tid] = u32 (elapsed) / (rep);
}

///////////////////////////////////////
__global__ void
kernel_gmem_cached_latency (u32* a, u32*b, u32*clock_count, int n)
{

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int t;
  clock_t start, elapsed = 0;

  int rep = 1;
  for (int i = 0; i < rep; i++)
    {
      start = clock ();
      t = b[tid];
      elapsed += clock () - start;
      a[tid] += t * (i + rep);
    }

  clock_count[tid] = u32 (elapsed) / (rep);
}

///////////////////////////////////////
__global__ void
kernel_gmem_departure_delay (u32* a, u32*b, u32*clock_count, int n, int d)
{

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int t;
  clock_t start, elapsed = 0;

  int rep = 10;
//  for (int i = 0; i < rep; i++)
    {
//      for (int j = 0; j < d; j++)
	{
//	  int idx=tid+j*n;

	  int idx = n + tid;
	  t = a[tid];
//	  elapsed += clock () - start;
	  b[tid] += t * (tid + rep);
	  start = clock ();
	  t = a[idx];
	  elapsed = clock () - start;
	  b[tid] += t * (tid + rep);
	}
    }

  clock_count[tid] = u32 (elapsed); // / (d*rep);
}

///////////////////////////////////////

int
compute_device_gmem_latency (u32 &LD_cached, u32 &LD_non_cached,
			     int n_log2 = 20, int block_size = 32)
{

  LD_cached = 0, LD_non_cached = 0;
  int n = (1 << n_log2);
  u32* host_a, *host_b, *host_clock;

  u32* dev_a, *dev_b, *dev_clock;
  host_a = (u32*) malloc (n * sizeof(u32));
  host_b = (u32*) malloc (n * sizeof(u32));
  host_clock = (u32*) malloc (n * sizeof(u32));

  for (int i = 0; i < n; i++)
    host_a[i] = 0xFFFFFFFF - i;
  for (int i = 0; i < n; i++)
    host_b[i] = 0xFFFFFFFF / 2 - (i / 2);
  memset (host_clock, 0x00, n * sizeof(u32));

  hipMalloc ((void**) &dev_a, n * sizeof(u32));
  hipMalloc ((void**) &dev_b, n * sizeof(u32));
  hipMalloc ((void**) &dev_clock, n * sizeof(u32));

  hipMemcpy (dev_a, host_a, n * sizeof(u32), hipMemcpyHostToDevice);
  hipMemcpy (dev_b, host_b, n * sizeof(u32), hipMemcpyHostToDevice);
  hipMemset (dev_clock, 0x00, n * sizeof(u32));
//  int block_size = 256;
//	printf("n=%d, n_blocks=%d, block_size=%d\n", n, n / block_size, block_size);
  kernel_gmem_cached_latency <<<n / block_size, block_size>>> (dev_a, dev_b,
							       dev_clock, n);
  hipMemcpy (host_clock, dev_clock, n * sizeof(u32), hipMemcpyDeviceToHost);

  u32 max_ld = 0;
  u32 min_ld = 0xFFFFFFFF;
  for (int i = 0; i < n; i++)
    {
      if (host_clock[i] > max_ld)
	max_ld = host_clock[i];
      if (host_clock[i] < min_ld)
	min_ld = host_clock[i];
    }
  LD_cached = max_ld;
//      printf("MAX_LD_CACHED =%d\n", max_ld);

  kernel_gmem_non_cached_latency <<<n / block_size, block_size>>> (dev_a, dev_b,
								   dev_clock,
								   n);
  hipMemcpy (host_clock, dev_clock, n * sizeof(u32), hipMemcpyDeviceToHost);
  max_ld = 0;
  min_ld = 0xFFFFFFFF;
  for (int i = 0; i < n; i++)
    {
      if (host_clock[i] > max_ld)
	max_ld = host_clock[i];
      if (host_clock[i] < min_ld)
	min_ld = host_clock[i];
    }
//    printf("MAX_LD_NON_CACHED =%d\n", max_ld);
  LD_non_cached = max_ld;

  hipFree (dev_a);
  hipFree (dev_b);
  hipFree (dev_clock);

  return 0;
}

///////////////////////////////////////

int
compute_device_departure_delay (u32 &departure_delay, int n_log2 = 20,
				int block_size = 32)
{

  departure_delay = 0;
  int n = (1 << n_log2);

  int d = 32;
  u32* host_a, *host_b, *host_clock;

  u32* dev_a, *dev_b, *dev_clock;

  host_a = (u32*) malloc (n * d * sizeof(u32));
  host_b = (u32*) malloc (n * sizeof(u32));
  host_clock = (u32*) malloc (n * sizeof(u32));

  for (int i = 0; i < n * d; i++)
    host_a[i] = 0xFFFFFFFF - i;
  for (int i = 0; i < n; i++)
    host_b[i] = 0xFFFFFFFF / 2 - (i / 2);
  memset (host_clock, 0x00, n * sizeof(u32));

  hipMalloc ((void**) &dev_a, n * d * sizeof(u32));
  hipMalloc ((void**) &dev_b, n * sizeof(u32));
  hipMalloc ((void**) &dev_clock, n * sizeof(u32));

  hipMemcpy (dev_a, host_a, n * d * sizeof(u32), hipMemcpyHostToDevice);
  hipMemcpy (dev_b, host_b, n * sizeof(u32), hipMemcpyHostToDevice);
  hipMemset (dev_clock, 0x00, n * sizeof(u32));
//  int block_size = 256;
//	printf("n=%d, n_blocks=%d, block_size=%d\n", n, n / block_size, block_size);
  kernel_gmem_departure_delay <<<n / block_size, block_size>>> (dev_a, dev_b,
								dev_clock, n,
								d);
  hipMemcpy (host_clock, dev_clock, n * sizeof(u32), hipMemcpyDeviceToHost);

  u32 max_ld = 0;
  u32 min_ld = 0xFFFFFFFF;
  for (int i = 0; i < n; i++)
    {
//      printf("[tid=%d, warp_id=%d, cc = %d] \n", i, (i/32), host_clock[i]);
//      printf("[tid=%-10d, warp_id=%-10d, cc = %lu] \n", i, (i/32), host_clock[i]);
      if (host_clock[i] > max_ld)
	max_ld = host_clock[i];
      if (host_clock[i] < min_ld)
	min_ld = host_clock[i];
    }

  for (int i = 0; i < n; i+=32)
    {
//      host_clock[i] -= min_ld;
      //      printf("[tid=%d, warp_id=%d, cc = %d] \n", i, (i/32), host_clock[i]);
      if (i % 32 == 0)
	printf ("[tid=%-10d, warp_id=%-10d, cc = %lu] \n", i, (i / 32),
		host_clock[i]);
    }

  departure_delay = max_ld;
//  LD_cached = max_ld;
//  printf ("DEPARTURE_DELAY = %d\n", max_ld);
//
//  kernel_gmem_non_cached_latency<<<n / block_size, block_size>>> (dev_a, dev_b,
//      dev_clock,
//      n);
//  hipMemcpy (host_clock, dev_clock, n * sizeof(u32), hipMemcpyDeviceToHost);
//  max_ld = 0;
//  min_ld = 0xFFFFFFFF;
//  for (int i = 0; i < n; i++)
//    {
//      if (host_clock[i] > max_ld)
//	max_ld = host_clock[i];
//      if (host_clock[i] < min_ld)
//	min_ld = host_clock[i];
//    }
//    printf("MAX_LD_NON_CACHED =%d\n", max_ld);
//  LD_non_cached = max_ld;
  return 0;
}

///////////////////////////////////////

//[Issue_cycles: 4]
//[Mem_bandwidth: 120]
//[Mem_LD: 420]
//[Departure_del_uncoal: 10]
//[Departure_del_coal: 4]
//[Active_SMs: 16]
//[Freq: 1]
//[Load_bytes_per_warp: 128]
int
host_get_device_specs_for_mwp_cwp (int argc, char ** argv)
{
//	printf("CUDA Device Query (Runtime API) version (CUDART static linking)\n");

  int n_devices = 0;
  hipError_t error_id = hipGetDeviceCount (&n_devices);

  if (error_id != hipSuccess)
    {
      printf ("hipGetDeviceCount returned %d\n-> %s\n", (int) error_id,
	      hipGetErrorString (error_id));
//		printf("Result = FAIL\n");
      exit (EXIT_FAILURE);
    }

// This function call returns 0 if there are no CUDA capable devices.
  if (n_devices == 0)
    {
      printf ("-There are no available device(s) that support CUDA\n");
      exit (EXIT_FAILURE);
    }
  else
    {
      printf ("[Found %d CUDA Capable device(s) ... ]\n", n_devices);
    }

  int dev_id, driver_version = 0, runtime_version = 0;

  for (dev_id = 0; dev_id < n_devices; ++dev_id)
    {
      char file_name[256];
      char device_name[256];

      hipSetDevice (dev_id);
      hipDeviceProp_t deviceProp;

      hipGetDeviceProperties (&deviceProp, dev_id);

      sprintf (device_name, "%s", deviceProp.name);
//		printf("checking device : %s\n", device_name);
      sprintf (file_name, "%s.specs", to_lower_str (device_name));
      printf ("writing specs to [%s] ... \n", file_name);
      FILE* file = fopen (file_name, "w");

      hipDriverGetVersion (&driver_version);
      hipRuntimeGetVersion (&runtime_version);
//				fprintf(file,
//				"(%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
//				deviceProp.multiProcessorCount,
//				_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
//				_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor)
//						* deviceProp.multiProcessorCount);
//		fprintf(file, "[max_clock_rate_khz: %d]\n", deviceProp.clockRate);
//		fprintf(file, "[max_clock_rate_mhz: %.0f]\n",
//				deviceProp.clockRate * 1e-3f);

      int memory_clock = 0;
      int memory_bus_width = 0;
      int l2_cache_size = 0;

#if CUDART_VERSION >= 5000
// This is supported in CUDA 5.0 (runtime API device properties)
//		printf("  Memory Clock rate:                             %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
//		printf("  Memory Bus Width:                              %d-bit\n", deviceProp.memoryBusWidth);

      memory_clock=deviceProp.memoryClockRate;
      memory_bus_width=deviceProp.memoryBusWidth;
      if (deviceProp.l2CacheSize)
	{
//			printf("  L2 Cache Size:                                 %d bytes\n", deviceProp.l2CacheSize);
	  l2_cache_size=deviceProp.l2CacheSize;
	}

#else
// This only available in CUDA 4.0-4.2 (but these were only exposed in the CUDA Driver API)
//		int memoryClock;
      getCudaAttribute<int> (&memory_clock,
			     hipDeviceAttributeMemoryClockRate, dev_id);

//		int memBusWidth;
      getCudaAttribute<int> (&memory_bus_width,
			     hipDeviceAttributeMemoryBusWidth,
			     dev_id);

//		int L2CacheSize;
      getCudaAttribute<int> (&l2_cache_size, hipDeviceAttributeL2CacheSize,
			     dev_id);

#endif

      u32 mem_ld = 0;
      u32 mem_ld_cached = 0;
      u64 max_mem_ld = 0;
      for (int b = 5; b <= 10; b++)
	{
	  int block_size = (1 << b);
	  for (int i = 10; i < 25; i++)
	    {
	      compute_device_gmem_latency (mem_ld_cached, mem_ld, i,
					   block_size);
	      if (mem_ld > max_mem_ld)
		max_mem_ld = mem_ld;
	    }
	}
//		mem_ld/=(20-10);
//		printf("mem_ld=%d\n",mem_ld);

      //??
      fprintf (file, "[Issue_cycles: %d]\n", 1);

      //checked.
      fprintf (
	  file,
	  "[Mem_bandwidth: %.2f]\n",
	  compute_peak_theoretical_bw_kflops (memory_bus_width, memory_clock)
	      * 1e-6f);

      //checked.
      fprintf (file, "[Mem_LD: %d]\n", max_mem_ld);
      //??
      fprintf (file, "[Departure_del_uncoal: %d]\n", 4);
      //??
      fprintf (file, "[Departure_del_coal: %d]\n", 4);

      //checked.
      fprintf (file, "[Active_SMs: %d]\n", deviceProp.multiProcessorCount);
      //checked.
      fprintf (file, "[Freq: %d]\n", int (deviceProp.clockRate * 1e-3f));
      //checked.
      fprintf (file, "[Load_bytes_per_warp: %d]\n", 128);

      fprintf (file, "\n");

//		fprintf(file, "[device_id: %d]\n", dev_id);
      fprintf (file, "[device_name: %s]\n", (device_name));
      fprintf (file, "[driver_version: %d.%d]\n", driver_version / 1000,
	       (driver_version % 100) / 10);
      fprintf (file, "[runtim_version: %d.%d]\n", runtime_version / 1000,
	       (runtime_version % 100) / 10);
      fprintf (file, "[compute_capability: %d.%d]\n", deviceProp.major,
	       deviceProp.minor);
      fprintf (file, "[global_memory_bytes: %llu]\n",
	       (unsigned long long) deviceProp.totalGlobalMem);
      fprintf (file, "[n_cores_per_sm: %3d]\n",
	       _ConvertSMVer2Cores (deviceProp.major, deviceProp.minor));

      fprintf (file, "[n_blocks_per_sm: %d]\n",
	       get_nblocks_per_SM (deviceProp.major, deviceProp.minor));

      fprintf (
	  file,
	  "[n_cores: %d]\n",
	  _ConvertSMVer2Cores (deviceProp.major, deviceProp.minor)
	      * deviceProp.multiProcessorCount);
      //		fprintf(file, "[memory_clock_rate_khz: %d]\n", memory_clock);
      //		fprintf(file, "[memory_clock_rate_mhz: %.0f]\n", memory_clock * 1e-3f);
      fprintf (file, "[memory_clock_rate_ghz: %.2f]\n", memory_clock * 1e-6f);

      fprintf (file, "[memory_bus_width_bits: %d]\n", memory_bus_width);
      if (l2_cache_size != 0)
	{
	  fprintf (file, "[l2_cache_size_bytes: %d]\n", l2_cache_size);
	}

//		printf(
//				"  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n",
//				deviceProp.maxTexture1D, deviceProp.maxTexture2D[0],
//				deviceProp.maxTexture2D[1], deviceProp.maxTexture3D[0],
//				deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
//		printf(
//				"  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
//				deviceProp.maxTexture1DLayered[0],
//				deviceProp.maxTexture1DLayered[1]);
//		printf(
//				"  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d layers\n",
//				deviceProp.maxTexture2DLayered[0],
//				deviceProp.maxTexture2DLayered[1],
//				deviceProp.maxTexture2DLayered[2]);

      fprintf (file, "[total_constant memory_bytes: %lu]\n",
	       deviceProp.totalConstMem);
      fprintf (file, "[total_shared_memory_per_block_bytes: %lu]\n",
	       deviceProp.sharedMemPerBlock);
      fprintf (file, "[total_registers_available_per_block: %d]\n",
	       deviceProp.regsPerBlock);
      fprintf (file, "[warp_size: %d]\n", deviceProp.warpSize);
      fprintf (file, "[maximum_number_of_threads_per_sm: %d]\n",
	       deviceProp.maxThreadsPerMultiProcessor);
      fprintf (file, "[maximum_number_of_threads_per_block: %d]\n",
	       deviceProp.maxThreadsPerBlock);
      fprintf (file, "[max_dimensions_of_block (x,y,z): (%d, %d, %d)]\n",
	       deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
	       deviceProp.maxThreadsDim[2]);
      fprintf (file, "[max_dimensions_of_grid (x,y,z): (%d, %d, %d)]\n",
	       deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
	       deviceProp.maxGridSize[2]);
//		fprintf(file,"  Maximum memory pitch:                          %lu bytes\n",
//				deviceProp.memPitch);
//		printf("  Texture alignment:                             %lu bytes\n",
//				deviceProp.textureAlignment);
      fprintf (file, "[concurrent_copy_and_kernel_execution: %d\n",
	       deviceProp.deviceOverlap);
      fprintf (file, "[n_copy_engines: %d]\n", deviceProp.asyncEngineCount);
//		fprintf(file,"  Run time limit on kernels:                     %s\n",
//				deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
      fprintf (file, "[integrated_gpu_sharing_host_memory: %d\n",
	       deviceProp.integrated);
      fprintf (file, "[support_host_page_locked_memory_mapping: %d]\n",
	       deviceProp.canMapHostMemory);
//		fprintf(file,"[alignment_requirement_for_surfaces: %d]\n",
//				deviceProp.surfaceAlignment);
      fprintf (file, "[ecc_support: %d]\n", deviceProp.ECCEnabled);
      fprintf (file, "[support_unified_addressing (UVA): %d]\n",
	       deviceProp.unifiedAddressing);
//		printf(
//				"  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n",
//				deviceProp.pciDomainID, deviceProp.pciBusID,
//				deviceProp.pciDeviceID);

//		const char *sComputeMode[] =
//				{
//						"Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
//						"Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
//						"Prohibited (no host thread can use ::hipSetDevice() with this device)",
//						"Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
//						"Unknown",
//						NULL };
//		printf("  Compute Mode:\n");
//		printf("     < %s >\n", sComputeMode[deviceProp.computeMode]);
      fclose (file);

      sprintf (file_name, "%s.mem", to_lower_str (device_name));
      printf ("writing mem load specs to [%s] ... \n", file_name);
      file = fopen (file_name, "w");

      u32 ld_cached, ld_noncached;
      for (int n = 10; n <= 20; n++)
	for (int block_size = 32; block_size <= 1024; block_size <<= 1)
	  {
	    compute_device_gmem_latency (ld_cached, ld_noncached, n,
					 block_size);

	    // log(n), block_size, cached_ld, non_cached_ld
	    fprintf (file, "[%d, %d, %d, %d]\n", n, block_size, ld_cached,
		     ld_noncached);
	  }
      fclose (file);
    }

// finish
// hipDeviceReset causes the driver to clean up all state. While
// not mandatory in normal operation, it is good practice.  It is also
// needed to ensure correct operation when the application is being
// profiled. Calling hipDeviceReset causes all profile data to be
// flushed before the application exits
  hipDeviceReset ();
  exit (EXIT_SUCCESS);
}

///////////////////////////////////////
int
test_device_gmem_latency (int argc, char ** argv)
{
  int n = 10;
  int block_size = 32;
  if (argc > 1)
    n = atoi (argv[1]);

  if (argc > 2)
    block_size = atoi (argv[2]);

  u32 ld_cached, ld_noncached;

  for (n = 10; n < 20; n++)
    for (block_size = 32; block_size <= 1024; block_size <<= 1)
      {
	compute_device_gmem_latency (ld_cached, ld_noncached, n, block_size);
	printf ("[n=2^%d, b=%d, cached=%d, non-cached=%d]\n", n, block_size,
		ld_cached, ld_noncached);
	printf ("============================\n");
      }
  return 0;
}

///////////////////////////////////////
int
test_device_gmem_departure_delay (int argc, char ** argv)
{
  int n = 10;
  int block_size = 32;
  if (argc > 1)
    n = atoi (argv[1]);

  if (argc > 2)
    block_size = atoi (argv[2]);
  u32 departure_delay = 0;
//  for (block_size = 32; block_size <= 1024; block_size <<= 1)
//    for (n = 10; n < 20; n++)
    {
      compute_device_departure_delay (departure_delay, n, block_size);
//	printf ("[n=2^%d, b=%d, departure_delay=%d]\n", n, block_size,
//		departure_delay);
//	printf ("============================\n");
    }
  return 0;
}

///////////////////////////////////////
// Program main
///////////////////////////////////////
int
main (int argc, char **argv)
{
//	return host_get_device_specs_full(argc, argv);
  return host_get_device_specs_for_mwp_cwp (argc, argv);
//	return compute_device_gmem_latency(argc, argv);

//  return test_device_gmem_latency (argc, argv);
//  return test_device_gmem_departure_delay(argc, argv);
}

///////////////////////////////////////

